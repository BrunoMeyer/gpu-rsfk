#include "hip/hip_runtime.h"
// Thrust includes
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/sort.h>


#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "matplotlibcpp.h"
namespace plt = matplotlibcpp;
 

#include <iostream> 
#include <stdio.h>
using namespace std; 
#include <cstdlib>
#include <cmath>
#include <bits/stdc++.h> 

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(1);
  }
}

#define HEAP_PARENT(i) ((i-1)/2)
#define HEAP_LEFT(i) ((2*i)+1)
#define HEAP_RIGHT(i) ((2*i)+2)

// #include <boost/preprocessor/repetition/repeat.hpp>


/*
PQ = Q-P

# Find equation hiperplane formula
H_bias = PQ.dot(Q)
H = PQ

# Verify if P is in the upper side of hiperplane
P_side = P.dot(H) > H_bias
*/


#define typepoints float
#define MAX_DEPTH 15

typedef struct TreeNode {
    // // A normal vector and a point of the hyperplane is sufficiently to represent it
    float* hyperplane_normal; // Difference between two points
    float* hyperplane_point; // Average of two points used to create a node
    
    // int skip_link;
    TreeNode* negative_path;
    TreeNode* positive_path;

    int leafs_count;
} TreeNode;


__device__
inline
void init_TreeNode(TreeNode* n, int node_index, TreeNode* tree, unsigned int* points_path, typepoints* points, int* actual_depth, int N, int D){
    n->leafs_count = 0;
    n->negative_path = nullptr;
    n->positive_path = nullptr;
    // n->negative_path = hyperplane_normal;
    // n->positive_path = hyperplane_point;
}

// __device__
// inline
// void init_TreeNode(TreeNode* n, int p1, int p2, int N, int D, typepoints* points){

// }


#define RANDOM_SEED 42


__global__
void test_random(){
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED, /* the seed controls the sequence of random values that are produced */
            blockIdx.x, /* the sequence number is only important with multiple cores */
            threadIdx.x, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            //   &states[blockIdx.x]);
            &r);
    printf("%d\n",hiprand(&r) % 100);
    printf("%d\n",hiprand(&r) % 100);
    printf("%d\n",hiprand(&r) % 100);
}

__global__
void test(typepoints* arr, int N, int D){
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED, /* the seed controls the sequence of random values that are produced */
              blockIdx.x, /* the sequence number is only important with multiple cores */
              threadIdx.x, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            //   &states[blockIdx.x]);
              &r);

    // arr[threadIdx.x*N+1] = N/2 + N/4;
    arr[threadIdx.x*D+1] = hiprand(&r) % N;
}

__device__
// inline
void tree_search(int point, TreeNode* node, TreeNode* tree, unsigned int* points_path, typepoints* points, int* actual_depth, int N, int D){
    
}

__device__
inline
void create_node(unsigned int node_idx, int p1, int p2, typepoints* tree, typepoints* points, int D)
{
    // Average point
    // node_path*D*2 : D*2 = size of centroid point and normal vector

    int i;
    // typepoints mean_axis_val;
    // typepoints normal_vector_axis_val;
    // typepoints plane_bias = 0;
    tree[node_idx*(D+1) + D] = 0.0f;

    for(i=0;i < D; ++i){
        // mean_axis_val = (points[p1*D+i]+points[p2*D+i])/2;
        // normal_vector_axis_val = points[p2*D+i]-points[p1*D+i];
        // tree[node_idx*(D+1) + i] = normal_vector_axis_val;
        // plane_bias+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
        tree[node_idx*(D+1) + i] = points[p1*D+i]-points[p2*D+i];
        tree[node_idx*(D+1) + D]+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
    }
    // tree[node_idx*(D+1) + D] = plane_bias;


    // if(node_idx == 2){
    //     printf("#########\n");
    //     for(i=0; i < D; ++i){
    //         printf("%f ", points[p1*D + i]);
    //     }
    //     printf("\n");
    //     for(i=0; i < D; ++i){
    //         printf("%f ", points[p2*D + i]);
    //     }
    //     printf("\n");
    //     for(i=0; i < D; ++i){
    //         printf("%f ", tree[node_idx*(D+1) + i]);
    //     }
    //     printf("\n");
    //     printf("%f\n", tree[node_idx*(D+1) + D]);
    //     printf("#########\n");
    // }
}


__device__
inline
int check_hyperplane_side(unsigned int node_idx, int p, typepoints* tree, typepoints* points, int D)
{
    typepoints aux = 0.0f;
    for(int i=0; i < D; ++i){
        aux += tree[node_idx*(D+1) + i]*points[p*D + i];
        // if(node_idx == 2){
        //     printf("\t%f ",aux);
        // }
    }
    // if(node_idx == 2){
    //     printf("\t>>> %f %f %f | %f %f %f | %f %f\n",tree[node_idx*(D+1)],tree[node_idx*(D+1)+1], tree[node_idx*(D+1)+2], points[p*D], points[p*D + 1], points[p*D + 2], aux, tree[node_idx*(D+1) + D]);
    // }
    
    return aux < tree[node_idx*(D+1) + D];
}


__global__
void build_tree(typepoints* tree, unsigned int* points_parent, bool* is_leaf, typepoints* points, int* actual_depth, int N, int D){
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED+threadIdx.x, /* the seed controls the sequence of random values that are produced */
            blockIdx.x, /* the sequence number is only important with multiple cores */
            threadIdx.x, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            //   &states[blockIdx.x]);
            &r);


    int p1, p2;

    // Sample two random points

    // int nodes_to_process = 1; // Root
    // int depth = 0;
    unsigned int node_thread = 0;
    // unsigned int node_idx = 0;
    int p;
    // unsigned int mask_depth = 2; // Start in second depth (second bit)
    
    

    if(threadIdx.x == 0){
        p1 = hiprand(&r) % N;
        p2 = p1;
        // Ensure that two different points was sampled
        while(p1 == p2 && N > 1){
            p2 = hiprand(&r) % N;
        }
        printf("Sampled points: %d %d\n", p1,p2);

        create_node(0, p1, p2, tree, points, D);
        *actual_depth = 1;
    }
    
    __syncthreads();
    
    

    while(*actual_depth < MAX_DEPTH){
        
        // Set nodes parent in the new depth
        for(p = threadIdx.x; p < N; p+=blockDim.x){
            // Heap left and right nodes are separeted by 1
            if(!is_leaf[points_parent[p]]){
                int right_child = check_hyperplane_side(points_parent[p], p, tree, points, D);
                // printf("\t%d %d <<\n", points_parent[p], right_child);
                points_parent[p] = HEAP_LEFT(points_parent[p])+right_child;

            }
        }
        __syncthreads();
        
        // if(threadIdx.x==0){
        //     for(unsigned int i=0; i < N; ++i){
        //         printf("\tpoint %d\ton cluster %d\n", i,points_parent[i]);
        //     }
        // }
        // __syncthreads();
        
        // Create new nodes
        if(threadIdx.x < pow(2,*actual_depth)){
            node_thread = 0; // start on root
            unsigned int bit_mask = 1;
            // Each thread find the node index to be created
            for(unsigned int i=1; i <= *actual_depth; ++i){
                // printf("%d %d %d << \n", ((threadIdx.x & bit_mask) != 0), threadIdx.x, bit_mask);
                node_thread = HEAP_LEFT(node_thread) + ((threadIdx.x & bit_mask) != 0);
                bit_mask = pow(2,i);
            }


            if(!is_leaf[HEAP_PARENT(node_thread)]){
                // if(threadIdx.x==0){
                //     for(unsigned int i=0; i < N; ++i){
                //         printf("%d\n", points_parent[i]);
                //     }
                // }
                // return;
                p1 = hiprand(&r) % N;
                int init_p_search = p1;
                while(points_parent[p1] != node_thread){
                    p1=((p1+1)%N);
                    if(p1 == init_p_search){
                        // printf("AAAAAAAAAAAAAAA\n");
                        p1 = -1;
                        break;
                    }
                    // printf("%d\n",p1);
                }
                p2 = hiprand(&r) % N;
                if(p1 == p2) p2=(p2+1)%N;

                // Ensure that two different points was sampled
                init_p_search = p2;
                while(p1 == p2  || points_parent[p2] != node_thread){
                    p2=((p2+1)%N);
                    if(p2 == init_p_search){
                        // printf("BBBBBBBBBBBBBBBB\n");
                        p2 = -1;
                        break;
                    }
                    // printf("%d\n",p2);

                }
                // printf("%d %d %d\n",node_thread, p1,p2);
                if(p1 != -1 && p2 != -1){
                    create_node(node_thread, p1, p2, tree, points, D);
                }
                else{
                    is_leaf[node_thread] = true;
                }
            }
            else{
                is_leaf[node_thread] = true;
            }
        }

        __syncthreads();

        if(threadIdx.x == 0){
            *actual_depth = *actual_depth+1;
            // printf("%d\n", *actual_depth);
        }
        __syncthreads();
        // return;

    }

    

    
    return;
}


int countDistinct(int* arr, int n) 
{ 
    // Creates an empty hashset 
    unordered_set<int> s; 
  
    // Traverse the input array 
    int res = 0; 
    for (int i = 0; i < n; i++) { 
  
        // If not present, then put it in 
        // hashtable and increment result 
        if (s.find(arr[i]) == s.end()) { 
            s.insert(arr[i]); 
            res++; 
        } 
    } 
  
    return res; 
} 

int main(int argc,char* argv[]) {
    // test_random<<<1,1>>>();
    // hipDeviceSynchronize();
    // return 0;

    // srand (time(NULL));
    srand(RANDOM_SEED);

    int N = atoi(argv[1]);
    int D = atoi(argv[2]);
    std::cout << N << std::endl;
    std::cout << D << std::endl;
    std::cout << std::endl;

    // thrust::copy(knn_indices, knn_indices + num_points * num_neighbors, knn_indices_device.begin());
    // thrust::device_vector<long> knn_indices_long_device(knn_indices_long, knn_indices_long + num_points * num_neighbors);   
    
    std::vector<typepoints> points(N*D);
    std::vector<int> labels(N*D);
    int total_labels = 2;

    for(int i=0; i < N; ++i){
        int l = rand() % N;
        // printf("%d ",i);
        for(int j=0; j < D; ++j){
            points[i*D+j] = l + (l>N/2)*N/2;
            // printf("%f ", points[i*D+j]);
        }
        // printf("\n");
        labels[i] = (l>N/2);
    }


    // std::vector<typepoints> X_axis(N);
    // std::vector<typepoints> Y_axis(N);
    
    // for(int i=0; i < N; ++i){
    //     X_axis[i] = points[i*D];
    //     Y_axis[i] = points[i*D + 1];
    // }
    // plt::scatter<typepoints,typepoints>(X_axis, Y_axis,10.0);
    // plt::show();

    thrust::device_vector<typepoints> device_points(points.begin(), points.end());
    // test<<<1,100>>>(thrust::raw_pointer_cast(device_points.data()), N, D);
    // thrust::copy(device_points.begin(), device_points.begin() + N*D, points.begin());

    // int MAX_NODES = 2*N-1 - N;
    int MAX_NODES = 0;
    for(int i=0; i < MAX_DEPTH; ++i){
        MAX_NODES+=pow(2,i);
    }
    

    std::cout << "MAX NODES: " << MAX_NODES << std::endl;
    // thrust::device_vector<TreeNode> device_tree(sizeof(TreeNode) * (MAX_NODES));
    thrust::device_vector<typepoints> device_tree((D + 1)*sizeof(typepoints) * (MAX_NODES));
    thrust::device_vector<bool> device_is_leaf(sizeof(bool) * MAX_NODES, false);
    thrust::device_vector<int> device_actual_depth(sizeof(int) * 1,0);
    thrust::device_vector<unsigned int> device_points_parent(sizeof(unsigned int) * N, 0);

    const int nt = 1024;
    auto t_start = std::chrono::high_resolution_clock::now();
    double elapsed_time_ms;

    build_tree<<<1,nt>>>(thrust::raw_pointer_cast(device_tree.data()),
                         thrust::raw_pointer_cast(device_points_parent.data()),
                         thrust::raw_pointer_cast(device_is_leaf.data()),
                         thrust::raw_pointer_cast(device_points.data()),
                         thrust::raw_pointer_cast(device_actual_depth.data()),
                         N, D);
    hipDeviceSynchronize();
    auto t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
    printf("Build Tree Kernel takes %lf seconds\n", elapsed_time_ms/1000);

    CudaTest("Build Tree Kernel failed!");   
    thrust::copy(device_points_parent.begin(), device_points_parent.begin() + N, labels.begin());

    total_labels = countDistinct(labels.data(),N);
    std::cout << "Total clusters: " << total_labels << std::endl;

    set<int> s; 
  
    for (int i = 0; i < N; i++) { 
        s.insert(labels[i]); 
    } 
    set<int>::iterator it; 
    
    // for (it = s.begin(); it != s.end(); ++it){
    //     int l = (int) *it; 
    //     int count_cluster = 0;
    //     for(int i=0; i < N; ++i){
    //         if(labels[i] == l) count_cluster++;
    //     }
    //     std::vector<typepoints> X_axis(count_cluster);
    //     std::vector<typepoints> Y_axis(count_cluster);
        
    //     int j =0;
    //     for(int i=0; i < N; ++i){
    //         if(labels[i] == l){
    //             X_axis[j] = points[i*D];
    //             Y_axis[j] = points[i*D + 1];
    //             ++j;
    //         }
    //     }
    //     plt::scatter<typepoints,typepoints>(X_axis, Y_axis,5.0,{
    //         {"alpha", "0.9"}
    //     });
    // }
    // plt::show();
}
