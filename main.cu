#include "hip/hip_runtime.h"
// Thrust includes
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/sort.h>


#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// #include <hip/hip_cooperative_groups.h>
// namespace cg = cooperative_groups;


#include "matplotlibcpp.h"
namespace plt = matplotlibcpp;

#include <iostream> 
#include <stdio.h>
using namespace std; 
#include <cstdlib>
#include <cmath>
#include <bits/stdc++.h> 

static void CudaTest(char* msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(1);
  }
}

#define HEAP_PARENT(i) ((i-1)/2)
#define HEAP_LEFT(i) ((2*i)+1)
#define HEAP_RIGHT(i) ((2*i)+2)

// #include <boost/preprocessor/repetition/repeat.hpp>


/*
PQ = Q-P

# Find equation hiperplane formula
H_bias = PQ.dot(Q)
H = PQ

# Verify if P is in the upper side of hiperplane
P_side = P.dot(H) > H_bias
*/


#define typepoints float
// #define MAX_DEPTH 11
#define RANDOM_SEED 42

typedef struct TreeNode {
    // // A normal vector and a point of the hyperplane is sufficiently to represent it
    float* hyperplane_normal; // Difference between two points
    float* hyperplane_point; // Average of two points used to create a node
    
    // int skip_link;
    TreeNode* negative_path;
    TreeNode* positive_path;

    int leafs_count;
} TreeNode;


__device__
inline
void init_TreeNode(TreeNode* n, int node_index, TreeNode* tree, unsigned int* points_path, typepoints* points, int* actual_depth, int N, int D){
    n->leafs_count = 0;
    n->negative_path = nullptr;
    n->positive_path = nullptr;
    // n->negative_path = hyperplane_normal;
    // n->positive_path = hyperplane_point;
}

// __device__
// inline
// void init_TreeNode(TreeNode* n, int p1, int p2, int N, int D, typepoints* points){

// }




__global__
void test_random(){
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED, /* the seed controls the sequence of random values that are produced */
            blockIdx.x, /* the sequence number is only important with multiple cores */
            threadIdx.x, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            //   &states[blockIdx.x]);
            &r);
    printf("%d\n",hiprand(&r) % 100);
    printf("%d\n",hiprand(&r) % 100);
    printf("%d\n",hiprand(&r) % 100);
}


#define MAX_K 1024
__global__
void
test_dynamic_vec_reg(int s){
    // register int* arr = new int[s];
    register int arr[MAX_K];
    for(int i=0; i < s; ++i) arr[i] = i*10; 
    for(int i=0; i < s; ++i) printf("%d\n",arr[i]); 
}

__global__
void test(typepoints* arr, int N, int D){
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED, /* the seed controls the sequence of random values that are produced */
              blockIdx.x, /* the sequence number is only important with multiple cores */
              threadIdx.x, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            //   &states[blockIdx.x]);
              &r);

    // arr[threadIdx.x*N+1] = N/2 + N/4;
    arr[threadIdx.x*D+1] = hiprand(&r) % N;
}

__device__
// inline
void tree_search(int point, TreeNode* node, TreeNode* tree, unsigned int* points_path, typepoints* points, int* actual_depth, int N, int D){
    
}

__device__
inline
void create_node(unsigned int node_idx, int p1, int p2, typepoints* tree, typepoints* points, int D)
{
    // Average point
    // node_path*D*2 : D*2 = size of centroid point and normal vector

    int i;
    // typepoints mean_axis_val;
    // typepoints normal_vector_axis_val;
    // typepoints plane_bias = 0;
    tree[node_idx*(D+1) + D] = 0.0f;

    for(i=0;i < D; ++i){
        // mean_axis_val = (points[p1*D+i]+points[p2*D+i])/2;
        // normal_vector_axis_val = points[p2*D+i]-points[p1*D+i];
        // tree[node_idx*(D+1) + i] = normal_vector_axis_val;
        // plane_bias+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
        tree[node_idx*(D+1) + i] = points[p1*D+i]-points[p2*D+i];
        tree[node_idx*(D+1) + D]+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
    }
    // tree[node_idx*(D+1) + D] = plane_bias;


    // if(node_idx == 2){
    //     printf("#########\n");
    //     for(i=0; i < D; ++i){
    //         printf("%f ", points[p1*D + i]);
    //     }
    //     printf("\n");
    //     for(i=0; i < D; ++i){
    //         printf("%f ", points[p2*D + i]);
    //     }
    //     printf("\n");
    //     for(i=0; i < D; ++i){
    //         printf("%f ", tree[node_idx*(D+1) + i]);
    //     }
    //     printf("\n");
    //     printf("%f\n", tree[node_idx*(D+1) + D]);
    //     printf("#########\n");
    // }
}


__device__
inline
int check_hyperplane_side(unsigned int node_idx, int p, typepoints* tree, typepoints* points, int D)
{
    typepoints aux = 0.0f;
    for(int i=0; i < D; ++i){
        aux += tree[node_idx*(D+1) + i]*points[p*D + i];
        // if(node_idx == 2){
        //     printf("\t%f ",aux);
        // }
    }
    // if(node_idx == 2){
    //     printf("\t>>> %f %f %f | %f %f %f | %f %f\n",tree[node_idx*(D+1)],tree[node_idx*(D+1)+1], tree[node_idx*(D+1)+2], points[p*D], points[p*D + 1], points[p*D + 2], aux, tree[node_idx*(D+1) + D]);
    // }
    
    return aux < tree[node_idx*(D+1) + D];
}

/*
__global__
void build_tree(typepoints* tree, unsigned int* points_parent, bool* is_leaf, typepoints* points, int* actual_depth, int N, int D){
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  the sequence number is only important with multiple cores 
            tid,  the offset is how much extra we advance in the sequence for each call, can be 0 
            //   &states[blockIdx.x]);
            &r);


    int p1, p2;
    // Sample two random points

    unsigned int node_thread = 0;
    int p;
    

    if(tid == 0){
        p1 = hiprand(&r) % N;
        p2 = p1;
        // Ensure that two different points was sampled
        while(p1 == p2 && N > 1){
            p2 = hiprand(&r) % N;
        }
        // printf("Sampled points: %d %d\n", p1,p2);

        create_node(0, p1, p2, tree, points, D);
        *actual_depth = 1;
    }
    

    __syncthreads();
    while(*actual_depth < MAX_DEPTH){
        
        // Set nodes parent in the new depth
        for(p = tid; p < N; p+=blockDim.x*gridDim.x){
            // Heap left and right nodes are separeted by 1
            if(!is_leaf[points_parent[p]]){
                int right_child = check_hyperplane_side(points_parent[p], p, tree, points, D);
                points_parent[p] = HEAP_LEFT(points_parent[p])+right_child;
            }
        }
        __syncthreads();
        
        // if(threadIdx.x==0){
        //     for(unsigned int i=0; i < N; ++i){
        //         printf("\tpoint %d\ton cluster %d\n", i,points_parent[i]);
        //     }
        // }
        // __syncthreads();
        
        // Create new nodes
        for(int nid=tid; nid < pow(2,*actual_depth); nid+=blockDim.x*gridDim.x){
            node_thread = 0; // start on root
            unsigned int bit_mask = 1;
            // Each thread find the node index to be created
            for(unsigned int i=1; i <= *actual_depth; ++i){
                node_thread = HEAP_LEFT(node_thread) + ((nid & bit_mask) != 0);
                bit_mask = pow(2,i);
            }


            if(!is_leaf[HEAP_PARENT(node_thread)]){
                p1 = hiprand(&r) % N;
                int init_p_search = p1;
                while(points_parent[p1] != node_thread){
                    p1=((p1+1)%N);
                    if(p1 == init_p_search){
                        p1 = -1;
                        break;
                    }
                }
                p2 = hiprand(&r) % N;
                if(p1 == p2) p2=(p2+1)%N;

                // Ensure that two different points was sampled
                init_p_search = p2;
                while(p1 == p2  || points_parent[p2] != node_thread){
                    p2=((p2+1)%N);
                    if(p2 == init_p_search){
                        p2 = -1;
                        break;
                    }
                }
                if(p1 != -1 && p2 != -1){
                    create_node(node_thread, p1, p2, tree, points, D);
                }
                else{
                    is_leaf[node_thread] = true;
                }
            }
            else{
                is_leaf[node_thread] = true;
            }
        }
        __syncthreads();

        if(threadIdx.x == 0){
            *actual_depth = *actual_depth+1;
        }
        __syncthreads();
    }
    return;
}
*/

__global__
void build_tree_init(typepoints* tree, unsigned int* points_parent, bool* is_leaf, typepoints* points, int* actual_depth, int N, int D){
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            //   &states[blockIdx.x]);
            &r);

    int p1, p2;
    // Sample two random points


    if(tid == 0){
        p1 = hiprand(&r) % N;
        p2 = p1;
        // Ensure that two different points was sampled
        while(p1 == p2 && N > 1){
            p2 = hiprand(&r) % N;
        }
        // printf("Sampled points: %d %d\n", p1,p2);

        create_node(0, p1, p2, tree, points, D);
        *actual_depth = 1;
    }
}


__global__
void build_tree_update_parents(typepoints* tree, unsigned int* points_parent, bool* is_leaf, typepoints* points, int* actual_depth, int N, int D){
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int right_child, p;

    // Set nodes parent in the new depth
    for(p = tid; p < N; p+=blockDim.x*gridDim.x){
        // Heap left and right nodes are separeted by 1
        if(!is_leaf[points_parent[p]]){
            right_child = check_hyperplane_side(points_parent[p], p, tree, points, D);
            points_parent[p] = HEAP_LEFT(points_parent[p])+right_child;
        }
    }
}

__global__
void build_tree_create_nodes(typepoints* tree, unsigned int* points_parent, bool* is_leaf, typepoints* points, int* actual_depth, int N, int D){
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            //   &states[blockIdx.x]);
            &r);

    int p1, p2;
    unsigned int bit_mask, i, node_thread;

    // Create new nodes
    for(int nid=tid; nid < pow(2,*actual_depth); nid+=blockDim.x*gridDim.x){
        node_thread = 0; // start on root
        bit_mask = 1;
        // Each thread find the node index to be created
        for(i=1; i <= *actual_depth; ++i){
            node_thread = HEAP_LEFT(node_thread) + ((nid & bit_mask) != 0);
            bit_mask = pow(2,i);
        }


        if(!is_leaf[HEAP_PARENT(node_thread)]){
            p1 = hiprand(&r) % N;
            int init_p_search = p1;
            while(points_parent[p1] != node_thread){
                p1=((p1+1)%N);
                if(p1 == init_p_search){
                    p1 = -1;
                    break;
                }
            }
            // p2 = hiprand(&r) % N;
            p2 = p1+1 % N;
            if(p1 == p2) p2=(p2+1)%N;

            // Ensure that two different points was sampled
            init_p_search = p2;
            while(p1 == p2  || points_parent[p2] != node_thread){
                p2=((p2+1)%N);
                if(p2 == init_p_search){
                    p2 = -1;
                    break;
                }
            }
            if(p1 != -1 && p2 != -1){
                create_node(node_thread, p1, p2, tree, points, D);
            }
            else{
                is_leaf[node_thread] = true;
            }
        }
        else{
            is_leaf[node_thread] = true;
        }
    }
    
}

__global__
void
build_tree_utils(int* actual_depth){
    *actual_depth = *actual_depth+1;
}



int countDistinct(int* arr, int n) 
{ 
    // Creates an empty hashset 
    unordered_set<int> s; 
  
    // Traverse the input array 
    int res = 0; 
    for (int i = 0; i < n; i++) { 
  
        // If not present, then put it in 
        // hashtable and increment result 
        if (s.find(arr[i]) == s.end()) { 
            s.insert(arr[i]); 
            res++; 
        } 
    } 
  
    return res; 
}

class Cron{
    public:
        std::chrono::time_point<std::chrono::high_resolution_clock> t_start;
        double t_total;

        Cron(){
            this->reset();
        }
        void reset(){
            t_total = 0.0;
            start();
        }
        void start(){
            t_start = std::chrono::high_resolution_clock::now();
        }
        double stop(){
            std::chrono::time_point<std::chrono::high_resolution_clock> t_end = std::chrono::high_resolution_clock::now();
            double dt = std::chrono::duration<double, std::milli>(t_end-t_start).count();
            t_total+=dt;
            return dt;
        }
};

int main(int argc,char* argv[]) {
    // test_random<<<1,1>>>();
    // test_dynamic_vec_reg<<<1,1>>>(15);
    // hipDeviceSynchronize();
    // return 0;

    // srand (time(NULL));
    srand(RANDOM_SEED);

    int N = atoi(argv[1]);
    int D = atoi(argv[2]);
    int MAX_DEPTH = atoi(argv[3]);
    int VERBOSE = atoi(argv[4]);

    std::cout << N << std::endl;
    std::cout << D << std::endl;
    std::cout << std::endl;

    // thrust::copy(knn_indices, knn_indices + num_points * num_neighbors, knn_indices_device.begin());
    // thrust::device_vector<long> knn_indices_long_device(knn_indices_long, knn_indices_long + num_points * num_neighbors);   
    
    std::vector<typepoints> points(N*D);
    std::vector<int> labels(N*D);
    int total_labels = 2;

    for(int i=0; i < N; ++i){
        int l = rand() % N;
        // printf("%d ",i);
        for(int j=0; j < D; ++j){
            // points[i*D+j] = l + (l>N/2)*N/2;
            points[i*D+j] = rand() % N;
            // printf("%f ", points[i*D+j]);
        }
        // printf("\n");
        labels[i] = (l>N/2);
    }


    // std::vector<typepoints> X_axis(N);
    // std::vector<typepoints> Y_axis(N);
    
    // for(int i=0; i < N; ++i){
    //     X_axis[i] = points[i*D];
    //     Y_axis[i] = points[i*D + 1];
    // }
    // plt::scatter<typepoints,typepoints>(X_axis, Y_axis,10.0);
    // plt::show();

    thrust::device_vector<typepoints> device_points(points.begin(), points.end());
    // test<<<1,100>>>(thrust::raw_pointer_cast(device_points.data()), N, D);
    // thrust::copy(device_points.begin(), device_points.begin() + N*D, points.begin());

    // int MAX_NODES = 2*N-1 - N;
    int MAX_NODES = 0;
    for(int i=0; i < MAX_DEPTH+1; ++i){
        MAX_NODES+=pow(2,i);
    }
    

    std::cout << "MAX NODES: " << MAX_NODES << std::endl;
    // thrust::device_vector<TreeNode> device_tree(sizeof(TreeNode) * (MAX_NODES));
    thrust::device_vector<typepoints> device_tree((D + 1)*sizeof(typepoints) * (MAX_NODES));
    thrust::device_vector<bool> device_is_leaf(sizeof(bool) * MAX_NODES, false);
    thrust::device_vector<int> device_actual_depth(sizeof(int) * 1,0);
    thrust::device_vector<unsigned int> device_points_parent(sizeof(unsigned int) * N, 0);

    const int nt = 1024;
    const int mp = 8;

    // build_tree<<<mp,nt>>>(thrust::raw_pointer_cast(device_tree.data()),
    //                      thrust::raw_pointer_cast(device_points_parent.data()),
    //                      thrust::raw_pointer_cast(device_is_leaf.data()),
    //                      thrust::raw_pointer_cast(device_points.data()),
    //                      thrust::raw_pointer_cast(device_actual_depth.data()),
    //                      N, D);
    // CudaTest("Build Tree Kernel failed!");

    build_tree_init<<<mp,nt>>>(thrust::raw_pointer_cast(device_tree.data()),
                               thrust::raw_pointer_cast(device_points_parent.data()),
                               thrust::raw_pointer_cast(device_is_leaf.data()),
                               thrust::raw_pointer_cast(device_points.data()),
                               thrust::raw_pointer_cast(device_actual_depth.data()),
                               N, D);

    if(VERBOSE >= 1){
        std::cout << std::endl;
    }

    Cron total_cron;
    Cron update_parents_cron;
    Cron create_nodes_cron;

    total_cron.start();
    for(int i=1; i < MAX_DEPTH; ++i){
        hipDeviceSynchronize();
        update_parents_cron.start();
        build_tree_update_parents<<<mp,nt>>>(thrust::raw_pointer_cast(device_tree.data()),
                                   thrust::raw_pointer_cast(device_points_parent.data()),
                                   thrust::raw_pointer_cast(device_is_leaf.data()),
                                   thrust::raw_pointer_cast(device_points.data()),
                                   thrust::raw_pointer_cast(device_actual_depth.data()),
                                   N, D);
        hipDeviceSynchronize();
        update_parents_cron.stop();
        CudaTest((char *)"build_tree_update_parents Kernel failed!");
        
        create_nodes_cron.start();
        build_tree_create_nodes<<<mp,nt>>>(thrust::raw_pointer_cast(device_tree.data()),
                                   thrust::raw_pointer_cast(device_points_parent.data()),
                                   thrust::raw_pointer_cast(device_is_leaf.data()),
                                   thrust::raw_pointer_cast(device_points.data()),
                                   thrust::raw_pointer_cast(device_actual_depth.data()),
                                   N, D);
        hipDeviceSynchronize();
        create_nodes_cron.stop();
        CudaTest((char *)"build_tree_create_nodes Kernel failed!");

        build_tree_utils<<<1,1>>>(thrust::raw_pointer_cast(device_actual_depth.data()));
        CudaTest((char *)"build_tree_utils Kernel failed!");

        if(VERBOSE >= 1){
            std::cout << "\e[ABuilding Tree Depth: " << i+1 << "/" << MAX_DEPTH << std::endl;
        }
    }
    total_cron.stop();
    hipDeviceSynchronize();
    
    if(VERBOSE >= 1){
        printf("Build Tree Kernel takes %lf seconds\n", create_nodes_cron.t_total/1000);
        printf("Update parents Kernel takes %lf seconds\n", update_parents_cron.t_total/1000);
        printf("Create nodes Kernel takes %lf seconds\n", create_nodes_cron.t_total/1000);
        thrust::copy(device_points_parent.begin(), device_points_parent.begin() + N, labels.begin());
        total_labels = countDistinct(labels.data(),N);
        std::cout << "Total clusters: " << total_labels << std::endl;
    }

    if(VERBOSE >= 2){

        set<int> s; 
    
        for (int i = 0; i < N; i++) { 
            s.insert(labels[i]); 
        } 
        set<int>::iterator it; 
        
        for (it = s.begin(); it != s.end(); ++it){
            int l = (int) *it; 
            int count_cluster = 0;
            for(int i=0; i < N; ++i){
                if(labels[i] == l) count_cluster++;
            }
            std::vector<typepoints> X_axis(count_cluster);
            std::vector<typepoints> Y_axis(count_cluster);
            
            int j =0;
            for(int i=0; i < N; ++i){
                if(labels[i] == l){
                    X_axis[j] = points[i*D];
                    Y_axis[j] = points[i*D + 1];
                    ++j;
                }
            }
            plt::scatter<typepoints,typepoints>(X_axis, Y_axis,5.0,{
                {"alpha", "0.9"}
            });
        }
        plt::show();
    }
}
