#include "hip/hip_runtime.h"
#ifndef __COMPUTE_KNN_FROM_BUCKETS__CU
#define __COMPUTE_KNN_FROM_BUCKETS__CU


__device__
inline
float euclidean_distance_sqr(typepoints* v1, typepoints* v2, int D)
{
    typepoints ret = 0.0f;
    typepoints diff;

    for(int i=0; i < D; ++i){
        diff = v1[i] - v2[i];
        ret += diff*diff;
    }

    return ret;
}

__global__
void compute_knn_from_buckets(int* points_parent,
                              int* points_depth,
                              int* accumulated_nodes_count,
                              int* child_count,
                              typepoints* points,
                              int* bucket_nodes,
                              int* knn_indices,
                              typepoints* knn_sqr_dist,
                              int N, int D, int max_bucket_size, int K,
                              int MAX_TREE_CHILD)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int parent_id, bucket_size, max_id_point, tmp_point;
    typepoints max_dist_val, tmp_dist_val;
    
    int knn_id;
    // extern __shared__ int local_knn_indices[];
    // extern __shared__ typepoints local_knn_sqr_dist[];
    
    // local_knn_indices = &local_knn_indices[MAX_TREE_CHILD*threadIdx.x];
    // local_knn_sqr_dist = &local_knn_sqr_dist[MAX_TREE_CHILD*threadIdx.x];

    for(int p = tid; p < N; p+=blockDim.x*gridDim.x){
        knn_id = p*K;
        __syncthreads();
        // __syncwarp();
        parent_id = accumulated_nodes_count[points_depth[p]] + points_parent[p];
        bucket_size = child_count[parent_id];
        // for(int i=0; i < K; ++i){
        //     local_knn_indices[i] = knn_indices[p*K+i];
        //     local_knn_sqr_dist[i] = knn_sqr_dist[p*K+i];
        // }

        // TODO: Run a first scan?
        max_id_point = knn_id;
        max_dist_val = knn_sqr_dist[knn_id];

        for(int j=1; j < K; ++j){
            if(knn_indices[knn_id+j] > max_dist_val){
                // tmp_dist_val = local_knn_sqr_dist[j];
                max_id_point = knn_id+j;
                max_dist_val = knn_sqr_dist[knn_id+j];
            }
        }
        
        for(int i=0; i < bucket_size; ++i){
            __syncthreads();
            tmp_point = bucket_nodes[max_bucket_size*parent_id + i];
            // if(p == tmp_point) continue;
            for(int j=0; j < K; ++j){
                if(tmp_point == knn_indices[knn_id+j]){
                    tmp_point = -1;
                    break;
                }
            }
            if(tmp_point == -1) continue;

            tmp_dist_val = euclidean_distance_sqr(&points[tmp_point*D], &points[p*D], D);

            if(tmp_dist_val < max_dist_val){
                // local_knn_indices[max_id_point] = tmp_point;
                // local_knn_sqr_dist[max_id_point] = tmp_dist_val;
                knn_indices[max_id_point] = tmp_point;
                knn_sqr_dist[max_id_point] = tmp_dist_val;

                max_dist_val = tmp_dist_val;
                for(int j=0; j < K; ++j){
                    // if(local_knn_sqr_dist[j] > max_dist_val){
                    if(knn_sqr_dist[knn_id+j] > max_dist_val){
                        max_id_point = knn_id+j;
                        // max_dist_val = local_knn_sqr_dist[j];
                        max_dist_val = knn_sqr_dist[knn_id+j];
                    }
                }
            }
        }
        // __syncthreads();
        // for(int i=0; i < K; ++i){
        //     knn_indices[p*K+i]  = local_knn_indices[i];
        //     knn_sqr_dist[p*K+i] = local_knn_sqr_dist[i];
        // }
    }
}

#endif