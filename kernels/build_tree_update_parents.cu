#include "hip/hip_runtime.h"
#ifndef __BUILD_TREE_UPDATE_PARENTS__CU
#define __BUILD_TREE_UPDATE_PARENTS__CU


__global__
void build_tree_update_parents(typepoints* tree,
                               int* tree_parents,
                               int* tree_children,
                               int* points_parent,
                               int* points_depth,
                               int* is_right_child,
                               bool* is_leaf,
                               int* sample_points,
                               int* child_count,
                               int* child_count_new_depth,
                               typepoints* points,
                               int* actual_depth,
                               int* tree_count,
                               int* depth_level_count,
                               int* count_new_nodes,
                               int N, int D)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    // hiprandState_t r; 
    // hiprand_init(RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
    //         blockIdx.x,  // the sequence number is only important with multiple cores 
    //         tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
    //         //   &states[blockIdx.x]);
    //         &r);

    int right_child, p;

    // Set nodes parent in the new depth
    for(p = tid; p < N; p+=blockDim.x*gridDim.x){
        // Heap left and right nodes are separeted by 1
        // printf(">%d %d\n", points_parent[p], is_leaf[points_parent[p]]);
        // if(is_leaf[points_parent[p]]){

        // TODO: Verify if the "if" statement is necessary
        if(points_depth[p] == *actual_depth-1 && child_count[points_parent[p]] > MAX_TREE_CHILD){
            right_child = is_right_child[p];
            // printf("DEBUG4: %d %d %d %d\n", points_parent[p], 2*points_parent[p]+right_child, tree_children[2*points_parent[p]+right_child], child_count[points_parent[p]]);
            // points_parent[p] = HEAP_LEFT(points_parent[p])+right_child;
            points_parent[p] = tree_children[2*points_parent[p]+right_child];
            // printf("%d %d %d\n", p, is_right_child[p], points_parent[p]);
            atomicAdd(&child_count_new_depth[points_parent[p]],1);
            points_depth[p] = *actual_depth;
        }
        // __syncwarp();
        // __syncthreads();
    }
}

__global__
void build_tree_post_update_parents(typepoints* tree,
                                    int* tree_parents,
                                    int* tree_children,
                                    int* points_parent,
                                    int* points_depth,
                                    int* is_right_child,
                                    bool* is_leaf,
                                    int* sample_points,
                                    int* child_count,
                                    typepoints* points,
                                    int* actual_depth,
                                    int* tree_count,
                                    int* depth_level_count,
                                    int* count_new_nodes,
                                    int N, int D)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    int p;

    
    // Set nodes parent in the new depth
    for(p = tid; p < N; p+=blockDim.x*gridDim.x){
        // device_sample_points[2*points_parent[p]] = p;
        // device_sample_points[2*points_parent[p]+1] = p;

        if(points_depth[p] == *actual_depth-1 && child_count[points_parent[p]] <= MAX_TREE_CHILD){
            is_leaf[points_parent[p]] = true;
            // printf("DEBUG5: %d\n", points_parent[p]);
        }

        // Heap left and right nodes are separeted by 1
        // if(device_sample_points[2*points_parent[p]]     == -1 &&
        //    device_sample_points[2*points_parent[p] + 1] != p){
        //         device_sample_points[2*points_parent[p]] = p;
        // }
        // if(device_sample_points[2*points_parent[p] + 1] == -1 &&
        //    device_sample_points[2*points_parent[p]]     != p){
        //         device_sample_points[2*points_parent[p]+1] = p;
        // }
    }
}

#endif