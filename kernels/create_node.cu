#include "hip/hip_runtime.h"
#ifndef __CREATE_NODE__CU
#define __CREATE_NODE__CU

__device__
inline
void create_root(typepoints* tree,
                 int* tree_parents,
                 int* tree_children,
                 int* tree_count,
                 int p1,
                 int p2,
                 typepoints* points,
                 int D)
{
    // Average point
    // node_path*D*2 : D*2 = size of centroid point and normal vector

    int node_idx = 0;
    
    tree_parents[node_idx] = -1;
    *tree_count = 0;

    int i;
    tree[node_idx*(D+1) + D] = 0.0f;

    for(i=0;i < D; ++i){
        tree[node_idx*(D+1) + i] = points[p1*D+i]-points[p2*D+i];
        tree[node_idx*(D+1) + D]+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
    }
}

__device__
inline
void create_node(int parent,
                 int is_right_child,
                 typepoints* tree,
                 int* tree_parents,
                 int* tree_children,
                 int* tree_count,
                 int* count_new_nodes,
                 int p1,
                 int p2,
                 typepoints* points,
                 int D)
{
    // Average point
    // node_path*D*2 : D*2 = size of centroid point and normal vector

    int node_idx = atomicAdd(tree_count, 1);
    // atomicAdd(count_new_nodes, 1);
    tree_parents[node_idx] = parent;
    
    tree_children[2*parent+is_right_child] = node_idx;
    // printf("DEBUG1: %d %d\n",2*parent+is_right_child, node_idx);
    int i;
    // typepoints mean_axis_val;
    // typepoints normal_vector_axis_val;
    // typepoints plane_bias = 0;
    tree[node_idx*(D+1) + D] = 0.0f;

    for(i=0; i < D; ++i){
        // mean_axis_val = (points[p1*D+i]+points[p2*D+i])/2;
        // normal_vector_axis_val = points[p2*D+i]-points[p1*D+i];
        // tree[node_idx*(D+1) + i] = normal_vector_axis_val;
        // plane_bias+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
        tree[node_idx*(D+1) + i] = points[p1*D+i]-points[p2*D+i];
        tree[node_idx*(D+1) + D]+= tree[node_idx*(D+1) + i]*(points[p1*D+i]+points[p2*D+i])/2; // multiply the point of plane and the normal vector 
    }
}

#endif