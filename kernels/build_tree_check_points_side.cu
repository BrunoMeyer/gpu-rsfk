#include "hip/hip_runtime.h"
#ifndef __BUILD_TREE_CHECK_POINTS_SIDE__CU
#define __BUILD_TREE_CHECK_POINTS_SIDE__CU


__device__
inline
int check_hyperplane_side(int node_idx, int p, typepoints* tree, typepoints* points, int D)
{
    typepoints aux = 0.0f;
    for(int i=0; i < D; ++i){
        aux += tree[node_idx*(D+1) + i]*points[p*D + i];
    }
    return aux < tree[node_idx*(D+1) + D];
}


__global__
void build_tree_check_points_side(typepoints* tree,
                                  int* tree_parents,
                                  int* tree_children,
                                  int* points_parent,
                                  int* points_depth,
                                  int* is_right_child,
                                  bool* is_leaf,
                                  int* sample_points,
                                  int* child_count,
                                  typepoints* points,
                                  int* actual_depth,
                                  int* tree_count,
                                  int* depth_level_count,
                                  int* accumulated_child_count,
                                  int* count_points_on_leafs,
                                  int* sample_candidate_points,
                                  int* points_id_on_sample,
                                  int N, int D, int RANDOM_SEED)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(*actual_depth*(RANDOM_SEED+blockDim.x)+RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            &r);


    int p, is_right;
    int csi; //candidate_sample_id;
    // Set nodes parent in the new depth
    for(p = tid; p < N; p+=blockDim.x*gridDim.x){
        // __syncwarp();
        // __syncthreads();
        
        if(points_depth[p] < *actual_depth-1 || is_leaf[points_parent[p]]) continue;
        
        // printf("%d %d\n", p, points_parent[p]);
        is_right = check_hyperplane_side(points_parent[p], p, tree, points, D);
        is_right_child[p] = is_right;
        // Threats to Validity: This assumes that all the follow properties are false:
        // - The atomic operations assumes an arbitrary/random order
        // - The points are shuffled

        // for(int is_right=0; is_right < 2; is_right++){
            if(sample_points[4*points_parent[p]   + 2*is_right] == -1){
                sample_points[4*points_parent[p]  + 2*is_right] = N;
            }
            // sample_points[4*points_parent[p] + 2*is_right + hiprand(&r) % 2] =  p;
            atomicMin(&sample_points[4*points_parent[p] + 2*is_right    ], p);
            atomicMax(&sample_points[4*points_parent[p] + 2*is_right + 1], p);
            
            // sample_points[4*points_parent[p] + 2*is_right + 0] =  1;
            // sample_points[4*points_parent[p] + 2*is_right + 1] =  1;

            // printf("%s: line %d: %d\n", __FILE__, __LINE__, 2*points_parent[p]+is_right);
            
            csi = atomicAdd(&count_points_on_leafs[2*points_parent[p]+is_right], 1);
            points_id_on_sample[p] = csi;
            // printf("%d\n",p);
            // if(accumulated_child_count[points_parent[p]]+csi > N) printf("%s: line %d : %d \n", __FILE__, __LINE__, accumulated_child_count[points_parent[p]]+csi);
            // sample_candidate_points[accumulated_child_count[points_parent[p]]+csi] = p;
        // }
    }
    // __syncthreads();
    // printf("%s: line %d : %d %d \n", __FILE__, __LINE__, depth_level_count[*actual_depth-1], *actual_depth);

}

#endif