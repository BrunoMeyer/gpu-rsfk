#include "hip/hip_runtime.h"
/*
This file is part of the GPU-RSFK Project (https://github.com/BrunoMeyer/gpu-rsfk).

BSD 3-Clause License

Copyright (c) 2021, Bruno Henrique Meyer, Wagner M. Nunan Zola
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef __BUILD_TREE_BUCKET_POINTS__CU
#define __BUILD_TREE_BUCKET_POINTS__CU

#include "../include/common.h"

__global__
void build_tree_bucket_points(int* points_parent,
                              int* points_depth,
                              int* accumulated_nodes_count,
                              int* node_idx_to_leaf_idx,
                              int* nodes_bucket,
                              int* bucket_sizes,
                              int N, int max_bucket_size, int total_leafs)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int my_id_on_bucket, parent_id;
    for(int p = tid; p < N; p+=blockDim.x*gridDim.x){
        parent_id = accumulated_nodes_count[points_depth[p]] + points_parent[p];
        my_id_on_bucket = atomicAdd(&bucket_sizes[node_idx_to_leaf_idx[parent_id]], 1);
        nodes_bucket[node_idx_to_leaf_idx[parent_id]*max_bucket_size + my_id_on_bucket] = p;
    }
}

#endif