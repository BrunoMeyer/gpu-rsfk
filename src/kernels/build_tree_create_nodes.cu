#include "hip/hip_runtime.h"
/*
This file is part of the GPU-RSFK Project (https://github.com/BrunoMeyer/gpu-rsfk).

BSD 3-Clause License

Copyright (c) 2021, Bruno Henrique Meyer, Wagner M. Nunan Zola
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef __BUILD_TREE_CREATE_NODES__CU
#define __BUILD_TREE_CREATE_NODES__CU

#include "../include/common.h"

__global__
void build_tree_create_nodes(RSFK_typepoints* tree_new_depth,
                             int* tree_parents_new_depth,
                             int* tree_children,
                             int* points_parent,
                             int* points_depth,
                             int* is_right_child,
                             bool* is_leaf,
                             bool* is_leaf_new_depth,
                             int* child_count,
                             RSFK_typepoints* points,
                             int* actual_depth,
                             int* tree_count,
                             int* depth_level_count,
                             int* count_new_nodes,
                             int* accumulated_nodes_count,
                             int* accumulated_child_count,
                             int* count_points_on_leafs,
                             int* sample_candidate_points,
                             int N, int D, int MIN_TREE_CHILD, int MAX_TREE_CHILD, int RANDOM_SEED)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int tidw = threadIdx.x % 32; // my id on warp
    
    hiprandState_t r; 
    hiprand_init(*actual_depth*(RANDOM_SEED+blockDim.x)+RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            &r);

    int p1, p2, node_thread;
    // int parent_id;
    int rand_id;
    
    int i, tmp_p1, tmp_p2, node_idx, tmp_node_idx, is_leaf_node;

    // Create new nodes
    // for(node_thread = tid; node_thread < depth_level_count[*actual_depth-1]; node_thread+=blockDim.x*gridDim.x){
    for(node_thread = tid; __any_sync(__activemask(), node_thread < depth_level_count[*actual_depth-1]); node_thread+=blockDim.x*gridDim.x){
        for(int is_right=0; is_right < 2; ++is_right){
            p1 = -1;
            is_leaf_node = 1;
            
            if(node_thread < depth_level_count[*actual_depth-1]){
                is_leaf_node = is_leaf[node_thread];
            }

            __syncthreads();
            if(!is_leaf_node){
            // if(!is_leaf[node_thread]){
                if(count_points_on_leafs[2*node_thread+is_right] > 0){
                    
                    rand_id = (hiprand(&r) % count_points_on_leafs[2*node_thread+is_right]);
                    p1 = sample_candidate_points[accumulated_child_count[2*node_thread+is_right]  +  rand_id];
                    rand_id = (hiprand(&r) % count_points_on_leafs[2*node_thread+is_right]);
                    p2 = sample_candidate_points[accumulated_child_count[2*node_thread+is_right]  +  rand_id];
                    
                    while(p1 == p2 && count_points_on_leafs[2*node_thread+is_right] >= MIN_TREE_CHILD){
                        rand_id = (hiprand(&r) % count_points_on_leafs[2*node_thread+is_right]);
                        p2 = sample_candidate_points[accumulated_child_count[2*node_thread+is_right]  + rand_id];
                    }

                    node_idx = atomicAdd(tree_count, 1);
                    tree_parents_new_depth[node_idx] = node_thread;
                    tree_children[2*node_thread+is_right] = node_idx;
                }
            }
            __syncthreads();
            // create_node(node_thread, is_right, tree_new_depth, tree_parents_new_depth,
            //             tree_children, tree_count, count_new_nodes, p1, p2, points, D, N);node_thread
            for(i=0; i < 32; ++i){
                tmp_p1 = __shfl_sync(__activemask(), p1, i);
                if(tmp_p1 == -1) continue;
                tmp_p2 = __shfl_sync(__activemask(), p2, i);
                tmp_node_idx = __shfl_sync(__activemask(), node_idx, i);
                // printf("%d %d %d\n", tmp_p1, tmp_2, tmp_node_idx)
                create_node(tree_new_depth, tree_parents_new_depth,
                            tree_children, tree_count, count_new_nodes, tmp_p1, tmp_p2, points, D, N, tmp_node_idx, tidw);
            }
        }
    }
}



/*
// TODO: SPLIT BY PROJECTION INTO A RANDOM DIRECTION

__global__
void init_random_directions(RSFK_typepoints* random_directions,
                            int random_directions_size,
                            int* actual_depth,
                            int RANDOM_SEED)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(*actual_depth*(RANDOM_SEED+blockDim.x)+RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            &r);

    for(int i = tid; i < random_directions_size; i+=blockDim.x*gridDim.x){
        random_directions[i] = 2*hiprand_uniform(&r) - 1.0f;
    }
}

__global__
void project_active_points(RSFK_typepoints* projection_values,
                           RSFK_typepoints* points,
                           int* active_points,
                           int* active_points_count,
                           int* points_parent,
                           int* is_right_child,
                           int* sample_candidate_points,
                           RSFK_typepoints* min_random_proj_values,
                           RSFK_typepoints* max_random_proj_values,
                           int N, int D)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int tidw = threadIdx.x % 32; // my id on warp

    int i,j,k;
    int p, tmp_p;
    int is_right;
    int node_idx, init_projection;

    __shared__ RSFK_typepoints product_threads[1024];
    int init_warp_on_block = threadIdx.x-tidw;

    for(i = tid; __any_sync(__activemask(), i < *active_points_count); i+=blockDim.x*gridDim.x){
        __syncthreads();
        
        p = -1;
        if(i < *active_points_count){
            p = active_points[i];
            is_right = is_right_child[p];
            node_idx = (2*points_parent[p]+is_right);
            init_projection = D*node_idx;
        }

        product_threads[init_warp_on_block + tidw] = 0.0f;
        
        __syncthreads();
        for(j=0; j < 32; ++j){
            tmp_p = __shfl_sync(__activemask(), p, j);
            if(tmp_p == -1) continue;
            __syncthreads();
            for(k=tidw; k < D; k+=32){
                product_threads[init_warp_on_block+j]+= projection_values[init_projection+k]*points[get_point_idx(tmp_p,k,N,D)];
            }
        }
        if(p == -1) continue;
        // atomicMin(&min_random_proj_values[node_idx],product_threads[init_warp_on_block+j]);
        // atomicMax(&max_random_proj_values[node_idx],product_threads[init_warp_on_block+j]);
    }
}

__global__
void choose_points_to_split(RSFK_typepoints* projection_values,
                           int* points_parent,
                           int* active_points,
                           int* active_points_count,
                           int* is_right_child,
                           int* sample_candidate_points,
                           RSFK_typepoints* min_random_proj_values,
                           RSFK_typepoints* max_random_proj_values,
                           int N, int D)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int tidw = threadIdx.x % 32; // my id on warp

    int i,j,k;
    int p, tmp_p;
    int is_right;
    int node_idx;

    __shared__ RSFK_typepoints product_threads[1024];
    int init_warp_on_block = threadIdx.x-tidw;

    for(i = tid; i < *active_points_count; i+=blockDim.x*gridDim.x){
        __syncthreads();
        
        p = active_points[i];
        is_right = is_right_child[p];
        node_idx = (2*points_parent[p]+is_right);

        // min_random_proj_values[node_idx]
        // atomicMax(&max_random_proj_values[node_idx],product_threads[init_warp_on_block+j]);
    }
}

__global__
void build_tree_create_nodes_random_projection(RSFK_typepoints* tree_new_depth,
                             int* tree_parents_new_depth,
                             int* tree_children,
                             int* points_parent,
                             int* points_depth,
                             int* is_right_child,
                             bool* is_leaf,
                             bool* is_leaf_new_depth,
                             int* child_count,
                             RSFK_typepoints* points,
                             int* actual_depth,
                             int* tree_count,
                             int* depth_level_count,
                             int* count_new_nodes,
                             int* accumulated_nodes_count,
                             int* accumulated_child_count,
                             int* count_points_on_leafs,
                             int* sample_candidate_points,
                             int* random_directions,
                             int N, int D, int K, int MAX_TREE_CHILD, int RANDOM_SEED)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(*actual_depth*(RANDOM_SEED+blockDim.x)+RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            &r);

    int p1, p2, node_thread;
    // int parent_id;
    int rand_id;

    float random_dir;
    int min_proj_point;
    
    // Create new nodes
    for(node_thread = tid; node_thread < depth_level_count[*actual_depth-1]; node_thread+=blockDim.x*gridDim.x){
        for(int is_right=0; is_right < 2; ++is_right){
            if(!is_leaf[node_thread]){
                if(count_points_on_leafs[2*node_thread+is_right] > 0){
                    random_dir = 2*hiprand_uniform(&r)-1.0f;
                    rand_id = (hiprand(&r) % count_points_on_leafs[2*node_thread+is_right]);
                    p1 = sample_candidate_points[accumulated_child_count[2*node_thread+is_right]  +  rand_id];
                    rand_id = (hiprand(&r) % count_points_on_leafs[2*node_thread+is_right]);
                    p2 = sample_candidate_points[accumulated_child_count[2*node_thread+is_right]  +  rand_id];
                    
                    while(p1 == p2 && count_points_on_leafs[2*node_thread+is_right] > K){
                        rand_id = (hiprand(&r) % count_points_on_leafs[2*node_thread+is_right]);
                        p2 = sample_candidate_points[accumulated_child_count[2*node_thread+is_right]  + rand_id];
                    }

                    __syncthreads();
                    create_node(node_thread, is_right, tree_new_depth, tree_parents_new_depth,
                                tree_children, tree_count, count_new_nodes, p1, p2, points, D, N);
                }
            }
        }
    }
}
*/
#endif