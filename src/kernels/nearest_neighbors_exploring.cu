#include "hip/hip_runtime.h"
#ifndef __NEAREST_NEIGHBORS_EXPLORING__CU
#define __NEAREST_NEIGHBORS_EXPLORING__CU

#include "../include/common.h"


__global__
void nearest_neighbors_exploring(typepoints* points,
                                 int* old_knn_indices,
                                 int* knn_indices,
                                 typepoints* knn_sqr_dist,
                                 int N, int D, int K)
{
    
    int p, tmp_p, tmp_candidate, max_id_point, p_neigh, p_neigh_neigh;
    int i,j,k;
    int knn_id;
    typepoints max_dist_val;

    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int lane = threadIdx.x % 32; // my id on warp
    
    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
    __shared__ typepoints candidate_dist_val[1024];
    int init_warp_on_block = threadIdx.x-lane;
    #else
    typepoints candidate_dist_val, tmp_candidate_dist_val;
    #endif



    for(p = tid; __any_sync(__activemask(), p < N); p+=blockDim.x*gridDim.x){
        if(p < N){
            knn_id = p*K;

            max_id_point = knn_id;
            max_dist_val = knn_sqr_dist[knn_id];


            for(j=1; j < K; ++j){
                if(knn_sqr_dist[knn_id+j] > max_dist_val){
                    max_id_point = knn_id+j;
                    max_dist_val = knn_sqr_dist[knn_id+j];
                }
            }
        }
        __syncwarp();
        for(i=0; i < K; ++i){
            if(p < N) p_neigh = old_knn_indices[knn_id+i];
            for(k=0; k < K; ++k){
                p_neigh_neigh = -1;

                if(p < N){
                    p_neigh_neigh = old_knn_indices[p_neigh*K + k];
                    
                    for(j=0; j < K; ++j){
                        if(p_neigh_neigh == knn_indices[knn_id+j]){
                            p_neigh_neigh = -1;
                            break;
                        }
                    }
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    candidate_dist_val[init_warp_on_block+lane] = 0.0f;
                    #else
                    candidate_dist_val = 0.0f;
                    #endif
                }

                __syncthreads();
                for(j=0; j < 32; ++j){
                    tmp_candidate = __shfl_sync(__activemask(), p_neigh_neigh, j);
                    if(tmp_candidate == -1) continue;
                    tmp_p = __shfl_sync(__activemask(), p, j);
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    euclidean_distance_sqr_coalesced(tmp_candidate, tmp_p, points, D, N,
                                                     lane, &candidate_dist_val[init_warp_on_block+j]);
                    #else
                    tmp_candidate_dist_val = euclidean_distance_sqr_coalesced(
                                                tmp_candidate, tmp_p, points, D, N,
                                                lane);
                    if(lane == j) candidate_dist_val = tmp_candidate_dist_val;
                    #endif
                }
                __syncwarp();
                if(p_neigh_neigh == -1) continue;

                #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                if(candidate_dist_val[init_warp_on_block+lane] < max_dist_val){
                #else
                if(candidate_dist_val < max_dist_val){
                #endif
                    // printf("%d %d\n", knn_indices[max_id_point] ,p_neigh_neigh);
                    knn_indices[max_id_point] = p_neigh_neigh;
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    knn_sqr_dist[max_id_point] = candidate_dist_val[init_warp_on_block+lane];
                    #else
                    knn_sqr_dist[max_id_point] = candidate_dist_val;
                    #endif

                    max_id_point = knn_id;
                    max_dist_val = knn_sqr_dist[knn_id];
                    for(j=1; j < K; ++j){
                        if(knn_sqr_dist[knn_id+j] > max_dist_val){
                            max_id_point = knn_id+j;
                            max_dist_val = knn_sqr_dist[knn_id+j];
                        }
                    }
                }
            }
        }
    }
}

#endif