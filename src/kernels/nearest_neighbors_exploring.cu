#include "hip/hip_runtime.h"
#ifndef __NEAREST_NEIGHBORS_EXPLORING__CU
#define __NEAREST_NEIGHBORS_EXPLORING__CU


__global__
void nearest_neighbors_exploring(typepoints* points,
                                 int* old_knn_indices,
                                 int* knn_indices,
                                 typepoints* knn_sqr_dist,
                                 int N, int D, int K)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    int max_id_point, p_neigh, p_neigh_neigh;
    typepoints max_dist_val, tmp_dist_val;
    
    int knn_id;

    for(int p = tid; p < N; p+=blockDim.x*gridDim.x){
        knn_id = p*K;
        // __syncthreads();
        // __syncwarp();

        max_id_point = knn_id;
        max_dist_val = knn_sqr_dist[knn_id];


        for(int j=1; j < K; ++j){
            if(knn_sqr_dist[knn_id+j] > max_dist_val){
                max_id_point = knn_id+j;
                max_dist_val = knn_sqr_dist[knn_id+j];
            }
        }
        
        for(int i=0; i < K; ++i){
            __syncthreads();
            p_neigh = old_knn_indices[knn_id+i];
            for(int k=0; k < K; ++k){
                p_neigh_neigh = old_knn_indices[p_neigh*K + k];
                if(p == p_neigh_neigh) continue;
                
                for(int j=0; j < K; ++j){
                    if(p_neigh_neigh == knn_indices[knn_id+j]){
                        p_neigh_neigh = -1;
                        break;
                    }
                }
                if(p_neigh_neigh == -1) continue;

                // tmp_dist_val = euclidean_distance_sqr(&points[p_neigh_neigh*D], &points[p*D], D);
                tmp_dist_val = euclidean_distance_sqr(p_neigh_neigh, p, points, D, N);

                if(tmp_dist_val < max_dist_val){
                    // printf("%d %d\n", knn_indices[max_id_point] ,p_neigh_neigh);
                    knn_indices[max_id_point] = p_neigh_neigh;
                    knn_sqr_dist[max_id_point] = tmp_dist_val;

                    max_id_point = knn_id;
                    max_dist_val = knn_sqr_dist[knn_id];
                    for(int j=1; j < K; ++j){
                        if(knn_sqr_dist[knn_id+j] > max_dist_val){
                            max_id_point = knn_id+j;
                            max_dist_val = knn_sqr_dist[knn_id+j];
                        }
                    }
                }
            }
        }
    }
}

__global__
void nearest_neighbors_exploring_coalesced(typepoints* points,
                                 int* old_knn_indices,
                                 int* knn_indices,
                                 typepoints* knn_sqr_dist,
                                 int N, int D, int K)
{
    
    int p, tmp_p, tmp_candidate, max_id_point, p_neigh, p_neigh_neigh;
    int i,j,k;
    int knn_id;
    typepoints max_dist_val;

    __shared__ typepoints candidate_dist_val[1024];

    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int tidw = threadIdx.x % 32; // my id on warp
    int init_warp_on_block = threadIdx.x-tidw;


    for(p = tid; __any_sync(__activemask(), p < N); p+=blockDim.x*gridDim.x){
        if(p < N){
            knn_id = p*K;
            // __syncthreads();
            // __syncwarp();

            max_id_point = knn_id;
            max_dist_val = knn_sqr_dist[knn_id];


            for(j=1; j < K; ++j){
                if(knn_sqr_dist[knn_id+j] > max_dist_val){
                    max_id_point = knn_id+j;
                    max_dist_val = knn_sqr_dist[knn_id+j];
                }
            }
        }
        __syncwarp();
        for(i=0; i < K; ++i){
            if(p < N) p_neigh = old_knn_indices[knn_id+i];
            for(k=0; k < K; ++k){
                p_neigh_neigh = -1;

                if(p < N){
                    p_neigh_neigh = old_knn_indices[p_neigh*K + k];
                    
                    for(j=0; j < K; ++j){
                        if(p_neigh_neigh == knn_indices[knn_id+j]){
                            p_neigh_neigh = -1;
                            break;
                        }
                    }
                    candidate_dist_val[init_warp_on_block+tidw] = 0.0f;
                }

                __syncthreads();
                // tmp_dist_val = euclidean_distance_sqr(&points[p_neigh_neigh*D], &points[p*D], D);
                for(j=0; j < 32; ++j){
                    tmp_candidate = __shfl_sync(__activemask(), p_neigh_neigh, j);
                    if(tmp_candidate == -1) continue;
                    tmp_p = __shfl_sync(__activemask(), p, j);
                    euclidean_distance_sqr_coalesced(tmp_candidate, tmp_p, points, D, N,
                                                     tidw, &candidate_dist_val[init_warp_on_block+j]);
                }
                __syncwarp();
                if(p_neigh_neigh == -1) continue;

                
                if(candidate_dist_val[init_warp_on_block+tidw] < max_dist_val){
                    // printf("%d %d\n", knn_indices[max_id_point] ,p_neigh_neigh);
                    knn_indices[max_id_point] = p_neigh_neigh;
                    knn_sqr_dist[max_id_point] = candidate_dist_val[init_warp_on_block+tidw];

                    max_id_point = knn_id;
                    max_dist_val = knn_sqr_dist[knn_id];
                    for(j=1; j < K; ++j){
                        if(knn_sqr_dist[knn_id+j] > max_dist_val){
                            max_id_point = knn_id+j;
                            max_dist_val = knn_sqr_dist[knn_id+j];
                        }
                    }
                }
            }
        }
    }
}

#endif