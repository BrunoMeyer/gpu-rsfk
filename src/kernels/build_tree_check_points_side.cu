#include "hip/hip_runtime.h"
#ifndef __BUILD_TREE_CHECK_POINTS_SIDE__CU
#define __BUILD_TREE_CHECK_POINTS_SIDE__CU

#include "../include/common.h"



__device__
inline
typepoints check_hyperplane_side(int node_idx, int p, typepoints* tree,
                                           typepoints* points, int D, int N,
                                           int* count_new_nodes,
                                           int tidw)
{
    typepoints s = 0.0f;
    for(int i=tidw; i < D; i+=32){
        s+=tree[get_tree_idx(node_idx,i,*count_new_nodes,D)]*points[get_point_idx(p,i,N,D)];
    }

    s += __shfl_xor_sync( 0xffffffff, s,  1); // assuming warpSize=32
    s += __shfl_xor_sync( 0xffffffff, s,  2); // assuming warpSize=32
    s += __shfl_xor_sync( 0xffffffff, s,  4); // assuming warpSize=32
    s += __shfl_xor_sync( 0xffffffff, s,  8); // assuming warpSize=32
    s += __shfl_xor_sync( 0xffffffff, s, 16); // assuming warpSize=32
    // lane 0 stores result in SHM
    return s;
}

__global__
void build_tree_check_active_points(int* points_parent,
                                    int* points_depth,
                                    bool* is_leaf,
                                    int* actual_depth,
                                    int* active_points,
                                    int* active_points_count,
                                    int N)
{
    int p;
    int pa_id;
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    for(p = tid; p < N; p+=blockDim.x*gridDim.x){
        if(points_depth[p] >= *actual_depth-1 && !is_leaf[points_parent[p]]){
            pa_id = atomicAdd(active_points_count, 1);
            active_points[pa_id] = p;
        }
    }
}

__global__
void build_tree_check_points_side(typepoints* tree,
                                  int* tree_parents,
                                  int* tree_children,
                                  int* points_parent,
                                  int* points_depth,
                                  int* is_right_child,
                                  bool* is_leaf,
                                  int* child_count,
                                  typepoints* points,
                                  int* actual_depth,
                                  int* tree_count,
                                  int* depth_level_count,
                                  int* accumulated_child_count,
                                  int* count_points_on_leafs,
                                  int* sample_candidate_points,
                                  int* points_id_on_sample,
                                  int* active_points,
                                  int* active_points_count,
                                  int* count_new_nodes,
                                  int N, int D, int RANDOM_SEED)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    int i, j, p, tmp_p;
    int csi;

    typepoints product, tmp_product; 

    int tidw = threadIdx.x % 32; // my id on warp

    // Set nodes parent in the new depth
    for(i = tid; __any_sync(__activemask(), i < *active_points_count); i+=blockDim.x*gridDim.x){
        p = -1;
        if(i < *active_points_count) p = active_points[i];
        
        for(j=0; j < 32; ++j){
            tmp_p = __shfl_sync(__activemask(), p, j);
            if(tmp_p == -1) continue;
            
            tmp_product = check_hyperplane_side(points_parent[tmp_p], tmp_p, tree, points, D, N,
                                                          count_new_nodes,
                                                          tidw);
                
            if(j == tidw) product = tmp_product;
        }
        __syncwarp();
        if(p == -1) continue;
        
        is_right_child[p] = product < tree[get_tree_idx(points_parent[p],D,*count_new_nodes,D)];
        
        csi = atomicAdd(&count_points_on_leafs[2*points_parent[p]+is_right_child[p]], 1);
        points_id_on_sample[p] = csi;
    }
}

#endif