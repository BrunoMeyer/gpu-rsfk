#include "hip/hip_runtime.h"
#ifndef __COMPUTE_KNN_FROM_BUCKETS__CU
#define __COMPUTE_KNN_FROM_BUCKETS__CU

#include "../include/common.h"

__device__
inline
float euclidean_distance_sqr(int p1, int p2, typepoints* points, int D, int N)
{
    typepoints ret = 0.0f;
    typepoints diff;

    for(int i=0; i < D; ++i){
        diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
        ret += diff*diff;
    }

    return ret;
}


__device__
inline
float euclidean_distance_sqr_small_block(int p1, int p2, typepoints* local_points,
                                         typepoints* points, int D, int N)
{
    typepoints ret = 0.0f;
    typepoints diff;

    for(int i=0; i < D; ++i){
        diff = local_points[get_point_idx(p1,i,32,D)] - points[get_point_idx(p2,i,N,D)];
        ret += diff*diff;
    }

    return ret;
}


__device__
inline
void euclidean_distance_sqr_coalesced_atomic(int p1, int p2, typepoints* points, int D,
                                             int N, int lane, typepoints* diff_sqd)
{
    typepoints diff;

    for(int i=lane; i < D; i+=32){
        diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
        atomicAdd(diff_sqd,diff*diff);
    }
}


#if EUCLIDEAN_DISTANCE_VERSION==EDV_ATOMIC_OK

   __device__
   inline
   void euclidean_distance_sqr_coalesced(int p1, int p2, typepoints* points, int D,
                                         int N, int lane, typepoints* diff_sqd)
   {
       typepoints diff;
       typepoints s = 0.0f;
       for(int i=lane; i < D; i+=32){
           diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
           s+=diff*diff;
           // atomicAdd(diff_sqd,diff*diff);
       }
       atomicAdd(diff_sqd,s);
   }

#elif EUCLIDEAN_DISTANCE_VERSION==EDV_ATOMIC_CSE  // common subexpression elimination

   __device__
   inline
   void euclidean_distance_sqr_coalesced(int p1, int p2, typepoints* points, int D,
                                         int N, int lane, typepoints* diff_sqd)
   {
       register typepoints diff;
       register typepoints s = 0.0f;
       // #define get_point_idx(point,dimension,N,D) (point*D+dimension)
       register typepoints* _p1 = &points[p1*D];
       register typepoints* _p2 = &points[p2*D];
       for(register int i=lane; i < D; i+=32){
           diff = _p1[i] - _p2[i];
           s+=diff*diff;
           // atomicAdd(diff_sqd,diff*diff);
       }
       atomicAdd(diff_sqd,s);
   }

#elif EUCLIDEAN_DISTANCE_VERSION==EDV_NOATOMIC

   __device__
   inline
   void euclidean_distance_sqr_coalesced(int p1, int p2, typepoints* points, int D,
                                         int N, int lane, typepoints* diff_sqd)
   {
       typepoints diff;
       typepoints s = 0.0f;
       for(int i=lane; i < D; i+=32){
           diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
           s+=diff*diff;
           // atomicAdd(diff_sqd,diff*diff);
       }
       //atomicAdd(diff_sqd,s);
       // do a simpler shfl_down warp reduce
       s += __shfl_down_sync( 0xffffffff, s, 16 );
       s += __shfl_down_sync( 0xffffffff, s,  8 );
       s += __shfl_down_sync( 0xffffffff, s,  4 );
       s += __shfl_down_sync( 0xffffffff, s,  2 );
       s += __shfl_down_sync( 0xffffffff, s,  1 );
       // lane 0 stores result in SHM
       if( threadIdx.x & 0x1f )
           *diff_sqd = s;
   }

#elif EUCLIDEAN_DISTANCE_VERSION==EDV_NOATOMIC_NOSHM

   __device__                // NOTE: value returned in register (NO SHM)
   inline                    // function return type CHANGED
   typepoints euclidean_distance_sqr_coalesced(int p1, int p2, typepoints* points, int D,
                                         int N, int lane)
   {
       typepoints diff;
       typepoints s = 0.0f;
       
       for(int i=lane; i < D; i+=32){
           diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
           s+=diff*diff;
           // atomicAdd(diff_sqd,diff*diff);
       }
       //atomicAdd(diff_sqd,s);
       // do a simpler shfl_down warp reduce
       s += __shfl_down_sync( 0xffffffff, s, 16 ); // assuming warpSize=32
       s += __shfl_down_sync( 0xffffffff, s,  8 ); // assuming warpSize=32
       s += __shfl_down_sync( 0xffffffff, s,  4 ); // assuming warpSize=32
       s += __shfl_down_sync( 0xffffffff, s,  2 ); // assuming warpSize=32
       s += __shfl_down_sync( 0xffffffff, s,  1 ); // assuming warpSize=32
       // broadcast reduced value to all threads in warp (so they can return the value)
       __shfl_down_sync( 0xffffffff, s,  1 );
       return s;
   }

#elif EUCLIDEAN_DISTANCE_VERSION==EDV_WARP_REDUCE_XOR

   __device__
   inline
   void euclidean_distance_sqr_coalesced(int p1, int p2, typepoints* points, int D,
                                         int N, int lane, typepoints* diff_sqd)
   {
       typepoints diff;
       typepoints s = 0.0f;
       for(int i=lane; i < D; i+=32){
           diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
           s+=diff*diff;
           // atomicAdd(diff_sqd,diff*diff);
       }
       //atomicAdd(diff_sqd,s);
       s += __shfl_xor_sync( 0xffffffff, s,  1); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s,  2); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s,  4); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s,  8); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s, 16); // assuming warpSize=32
       // lane 0 stores result in SHM
       if( threadIdx.x & 0x1f )
           *diff_sqd = s;
   }

#elif EUCLIDEAN_DISTANCE_VERSION==EDV_WARP_REDUCE_XOR_NOSHM

   __device__                // NOTE: value returned in register (NO SHM)
   inline                    // function return type CHANGED
   typepoints euclidean_distance_sqr_coalesced(int p1, int p2, typepoints* points, int D,
                                         int N, int lane)
   {
       typepoints diff;
       typepoints s = 0.0f;
       
       for(int i=lane; i < D; i+=32){
            diff = points[get_point_idx(p1,i,N,D)] - points[get_point_idx(p2,i,N,D)];
            s+=diff*diff;
       }
       //atomicAdd(diff_sqd,s);
       s += __shfl_xor_sync( 0xffffffff, s,  1); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s,  2); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s,  4); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s,  8); // assuming warpSize=32
       s += __shfl_xor_sync( 0xffffffff, s, 16); // assuming warpSize=32
       // all lanes have the value, just return it
       return s;
   }

#endif


// Assign a bucket (leaf in the tree) to each warp and a point to each thread (persistent kernel)
__global__
void compute_knn_from_buckets_perwarp_coalesced(int* points_parent,
                              int* points_depth,
                              int* accumulated_nodes_count,
                              typepoints* points,
                              int* node_idx_to_leaf_idx,
                              int* nodes_bucket,
                              int* bucket_size,
                              int* knn_indices,
                              typepoints* knn_sqr_dist,
                              int N, int D, int max_bucket_size, int K,
                              int MAX_TREE_CHILD, int total_buckets)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int parent_id, current_bucket_size, max_id_point, candidate_point;
    typepoints max_dist_val;
    
    int knn_id;
    int lane = threadIdx.x % 32; // my id on warp
    

    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
    __shared__ typepoints candidate_dist_val[1024];
    int init_warp_on_block = (threadIdx.x/32)*32;
    #else
    typepoints candidate_dist_val, tmp_dist_val;
    #endif

    int bid, p, _p, i, j;
    int tmp_candidate, tmp_p;
    
    for(bid = tid/32; bid < total_buckets; bid+=blockDim.x*gridDim.x/32){
        current_bucket_size = bucket_size[bid];
        for(_p = lane; __any_sync(__activemask(),_p < current_bucket_size); _p+=32){
            if(_p < current_bucket_size){
                p = nodes_bucket[bid*max_bucket_size + _p];
                knn_id = p*K;
                parent_id = accumulated_nodes_count[points_depth[p]] + points_parent[p];
                current_bucket_size = bucket_size[node_idx_to_leaf_idx[parent_id]];

                max_id_point = knn_id;
                max_dist_val = knn_sqr_dist[knn_id];
                // Finds the index of the furthest point from the current result of knn_indices
                // and the distance between them
                for(j=1; j < K; ++j){
                    if(knn_sqr_dist[knn_id+j] > max_dist_val){
                        max_id_point = knn_id+j;
                        max_dist_val = knn_sqr_dist[knn_id+j];
                    }
                }
            }
            for(i=0; i < current_bucket_size; ++i){

                candidate_point = -1;
                if(_p < current_bucket_size){
                    candidate_point = nodes_bucket[node_idx_to_leaf_idx[parent_id]*max_bucket_size + i];
                    
                    // Verify if the candidate point (inside the bucket of current point)
                    // already is in the knn_indices result
                    for(j=0; j < K; ++j){
                        if(candidate_point == knn_indices[knn_id+j]){
                            candidate_point = -1;
                            break;
                        }
                    }
                    // If it is, then it doesnt need to be treated, then go to
                    // the next iteration and wait the threads from same warp to goes on
                }

                #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                candidate_dist_val[threadIdx.x] = 0.0f;
                #endif

                for(j=0; j < 32; ++j){
                    __syncwarp();
                    tmp_candidate = __shfl_sync(0xffffffff, candidate_point, j);
                    if(tmp_candidate == -1) continue;
                    tmp_p = __shfl_sync(0xffffffff, p, j);
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    euclidean_distance_sqr_coalesced(tmp_candidate, tmp_p, points, D, N,
                                                    lane,
                                                    &candidate_dist_val[init_warp_on_block+j]);
                    #else
                    tmp_dist_val = euclidean_distance_sqr_coalesced(tmp_candidate, tmp_p, points, D, N, lane);
                    if(lane == j) candidate_dist_val = tmp_dist_val;
                    #endif
                }
                if(candidate_point == -1) continue;

                // If the candidate is closer than the pre-computed furthest point,
                // switch them
                #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                if(candidate_dist_val[threadIdx.x] < max_dist_val){
                #else
                if(candidate_dist_val < max_dist_val){
                #endif
                    knn_indices[max_id_point] = candidate_point;
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    knn_sqr_dist[max_id_point] = candidate_dist_val[threadIdx.x];
                    #else
                    knn_sqr_dist[max_id_point] = candidate_dist_val;
                    #endif
                    // Also update the furthest point that will be used in the next
                    // comparison
                    max_id_point = knn_id;
                    max_dist_val = knn_sqr_dist[knn_id];
                    for(j=1; j < K; ++j){
                        if(knn_sqr_dist[knn_id+j] > max_dist_val){
                            max_id_point = knn_id+j;
                            max_dist_val = knn_sqr_dist[knn_id+j];
                        }
                    }
                }
            }
        }
    }
}

// Assign a bucket (leaf in the tree) to each block (persistent kernel)
// In this kernel, the redundant computation of symetric distances is avoided

// Since different points neighborhood may be updated by different threads,
// a lock system must be implemented
__global__
void compute_knn_from_buckets_perblock_coalesced_symmetric(int* points_parent,
                              int* points_depth,
                              int* accumulated_nodes_count,
                              typepoints* points,
                              int* node_idx_to_leaf_idx,
                              int* nodes_bucket,
                              int* bucket_size,
                              int* knn_indices,
                              typepoints* knn_sqr_dist,
                              int N, int D, int max_bucket_size, int K,
                              int MAX_TREE_CHILD, int total_buckets)
{
    int cbs; // cbs = current bucket size
    
    int knn_id;
    int wid = threadIdx.x / 32; // my id on warp
    int lane = threadIdx.x % 32; // my id on warp
    
    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
    __shared__ typepoints candidate_dist_val[32];
    #else
    typepoints candidate_dist_val;
    #endif

    int bid, p1, p2, real_p1, real_p2, _p, i, j;
    
    __shared__ int sm_leaf_bucket[300];
    __shared__ typepoints max_dist_val[300];
    __shared__ int max_position[300];

    int done_p1, done_p2;
    __shared__ int lock_point[300];
    
    bid = blockIdx.x;
    cbs = bucket_size[bid];
    for(i=threadIdx.x; i < cbs; i+=blockDim.x){
        p1 = nodes_bucket[bid*max_bucket_size + i];
        sm_leaf_bucket[i] = p1;
        lock_point[i] = 0;

        knn_id = p1*K;

        max_position[i] = knn_id;
        max_dist_val[i] = knn_sqr_dist[knn_id];
        // Finds the index of the furthest point from the current result of knn_indices
        // and the distance between them
        for(j=1; j < K; ++j){
            if(knn_sqr_dist[knn_id+j] > max_dist_val[i]){
                max_position[i] = knn_id+j; // The initial point is not necessarily in the bucket
                max_dist_val[i] = knn_sqr_dist[knn_id+j];
            }
        }
    }

    __syncthreads();
    
    for(_p = wid; _p < (cbs*cbs - cbs)/2; _p+=blockDim.x/32){
        p1 = cbs - 2 - floor(sqrt((float)((-8*_p + 4*cbs*(cbs-1)-7)))/2.0 - 0.5);
        p2 = _p + p1 + 1 - cbs*(cbs-1)/2 + (cbs-p1)*((cbs-p1)-1)/2;
        real_p1 = sm_leaf_bucket[p1];
        real_p2 = sm_leaf_bucket[p2];

        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        candidate_dist_val[wid] = 0.0f;
        #endif

        __syncwarp();
        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        euclidean_distance_sqr_coalesced(real_p1,
                                         real_p2,
                                         points, D, N,
                                         lane,
                                         &candidate_dist_val[wid]);
        #else
        candidate_dist_val = euclidean_distance_sqr_coalesced(real_p1,
                                                              real_p2,
                                                              points, D, N, lane);
        #endif
        __syncwarp();
        
        if(lane == 0){
            #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
            done_p1 = candidate_dist_val[wid] >= max_dist_val[p1];
            done_p2 = candidate_dist_val[wid] >= max_dist_val[p2];
            #else
            done_p1 = candidate_dist_val >= max_dist_val[p1];
            done_p2 = candidate_dist_val >= max_dist_val[p2];
            #endif

            for(j=0; j < K && (!done_p1 || !done_p2); ++j){
                done_p1 |= real_p2 == knn_indices[real_p1*K+j];
                done_p2 |= real_p1 == knn_indices[real_p2*K+j];
            }

            while(!done_p1 || !done_p2){
                if(!done_p1 && !atomicCAS(&lock_point[p1], 0, 1)){
                    done_p1 = 1;
                    // If the candidate is closer than the pre-computed furthest point,
                    // switch them
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    if(candidate_dist_val[wid] < max_dist_val[p1]){
                    #else
                    if(candidate_dist_val < max_dist_val[p1]){
                    #endif
                        knn_indices[max_position[p1]] = real_p2;
                        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                        knn_sqr_dist[max_position[p1]] = candidate_dist_val[wid];
                        #else
                        knn_sqr_dist[max_position[p1]] = candidate_dist_val;
                        #endif

                        // Also update the furthest point that will be used in the next
                        // comparison
                        knn_id = real_p1*K;
                        max_position[p1] = knn_id;
                        max_dist_val[p1] = knn_sqr_dist[knn_id];
                        for(j=1; j < K; ++j){
                            if(knn_sqr_dist[knn_id+j] > max_dist_val[p1]){
                                max_position[p1] = knn_id+j;
                                max_dist_val[p1] = knn_sqr_dist[knn_id+j];
                            }
                        }
                    }
                    atomicExch(&lock_point[p1], 0);
                }

                if(!done_p2 && !atomicCAS(&lock_point[p2], 0, 1)){
                    done_p2 = 1;
                    // If the candidate is closer than the pre-computed furthest point,
                    // switch them
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    if(candidate_dist_val[wid] < max_dist_val[p2]){
                    #else
                    if(candidate_dist_val < max_dist_val[p2]){
                    #endif
                        knn_indices[max_position[p2]] = real_p1;
                        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                        knn_sqr_dist[max_position[p2]] = candidate_dist_val[wid];
                        #else
                        knn_sqr_dist[max_position[p2]] = candidate_dist_val;
                        #endif
        
                        // Also update the furthest point that will be used in the next
                        // comparison
                        knn_id = real_p2*K;
                        max_position[p2] = knn_id;
                        max_dist_val[p2] = knn_sqr_dist[knn_id];
                        for(j=1; j < K; ++j){
                            if(knn_sqr_dist[knn_id+j] > max_dist_val[p2]){
                                max_position[p2] = knn_id+j;
                                max_dist_val[p2] = knn_sqr_dist[knn_id+j];
                            }
                        }
                    }
                    atomicExch(&lock_point[p2], 0);
                }
            }
        }
        __syncwarp();
    }
    
}


// This kernel is a optmization of compute_knn_from_buckets_perblock_coalesced_symmetric kernel
// The optimization consists use and communicate idle threads during lock system
__global__
void compute_knn_from_buckets_perblock_coalesced_symmetric_dividek(
                              typepoints* points,
                              int* nodes_bucket,
                              int* bucket_size,
                              int* knn_indices,
                              typepoints* knn_sqr_dist,
                              int N, int D, int max_bucket_size, int K,
                              int MAX_TREE_CHILD, int total_buckets)
{
    int cbs; // cbs = current bucket size
    
    int knn_id;
    int wid = threadIdx.x / 32; // my warp id
    int lane = threadIdx.x % 32; // my id on warp
    

    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
    __shared__ typepoints candidate_dist_val[32];
    #else
    typepoints candidate_dist_val;
    #endif

    int bid, p1, p2, real_p1, real_p2, _p, i, j;
    
    __shared__ int sm_leaf_bucket[1024];
    __shared__ typepoints max_dist_val[1024];
    __shared__ int max_position[1024];
    
    int local_max_position, tmp_max_position;
    typepoints local_max_dist, tmp_max_dist;

    int done_p1, done_p2;
    __shared__ int lock_point[1024];

    // Non persistent kernel seems to be more efficient
    bid=blockIdx.x;
    __syncthreads();
    cbs = bucket_size[bid];
    for(i=threadIdx.x; i < cbs; i+=blockDim.x){
        p1 = nodes_bucket[bid*max_bucket_size + i];
        sm_leaf_bucket[i] = p1;
        lock_point[i] = 0;

        knn_id = p1*K;

        max_position[i] = knn_id;
        max_dist_val[i] = knn_sqr_dist[knn_id];
        // Finds the index of the furthest point from the current result of knn_indices
        // and the distance between them
        for(j=1; j < K; ++j){
            if(knn_sqr_dist[knn_id+j] > max_dist_val[i]){
                max_position[i] = knn_id+j; // The initial point is not necessarily in the bucket
                max_dist_val[i] = knn_sqr_dist[knn_id+j];
            }
        }

    }

    __syncthreads();
    
    for(_p = wid; _p < (cbs*cbs - cbs)/2; _p+=blockDim.x/32){
        p1 = cbs - 2 - floor(sqrt((float)((-8*_p + 4*cbs*(cbs-1)-7)))/2.0 - 0.5);
        p2 = _p + p1 + 1 - cbs*(cbs-1)/2 + (cbs-p1)*((cbs-p1)-1)/2;
        real_p1 = sm_leaf_bucket[p1];
        real_p2 = sm_leaf_bucket[p2];
        // __syncwarp();


        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        candidate_dist_val[wid] = 0.0f;
        #endif

        // __syncwarp();
        // __syncthreads();
        // __syncwarp();
        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        euclidean_distance_sqr_coalesced(real_p1,
                                        real_p2,
                                        points, D, N,
                                        lane,
                                        &candidate_dist_val[wid]);
        __syncwarp();
        #else
        candidate_dist_val = euclidean_distance_sqr_coalesced(real_p1,
                                                            real_p2,
                                                            points, D, N, lane);
        #endif
        
        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        done_p1 = candidate_dist_val[wid] >= max_dist_val[p1];
        done_p2 = candidate_dist_val[wid] >= max_dist_val[p2];
        #else
        done_p1 = candidate_dist_val >= max_dist_val[p1];
        done_p2 = candidate_dist_val >= max_dist_val[p2];
        #endif
        
        // Verify if the candidate point already is in the knn_indices
        for(j=lane; j < K && (!done_p1 || !done_p2); j+=32){
            done_p1 |= real_p2 == knn_indices[real_p1*K+j];
            done_p2 |= real_p1 == knn_indices[real_p2*K+j];
        }
        
        done_p1 |= __shfl_xor_sync( 0xffffffff, done_p1,  1); // assuming warpSize=32
        done_p1 |= __shfl_xor_sync( 0xffffffff, done_p1,  2); // assuming warpSize=32
        done_p1 |= __shfl_xor_sync( 0xffffffff, done_p1,  4); // assuming warpSize=32
        done_p1 |= __shfl_xor_sync( 0xffffffff, done_p1,  8); // assuming warpSize=32
        done_p1 |= __shfl_xor_sync( 0xffffffff, done_p1, 16); // assuming warpSize=32
        
        done_p2 |= __shfl_xor_sync( 0xffffffff, done_p2,  1); // assuming warpSize=32
        done_p2 |= __shfl_xor_sync( 0xffffffff, done_p2,  2); // assuming warpSize=32
        done_p2 |= __shfl_xor_sync( 0xffffffff, done_p2,  4); // assuming warpSize=32
        done_p2 |= __shfl_xor_sync( 0xffffffff, done_p2,  8); // assuming warpSize=32
        done_p2 |= __shfl_xor_sync( 0xffffffff, done_p2, 16); // assuming warpSize=32

        
        
        while(!done_p1 || !done_p2){
            // if(!done_p1 && !atomicOr(&lock_point[p1], 1)){
            if(!done_p1 && __any_sync(0xffffffff,!atomicCAS(&lock_point[p1], 0, 1))){
                done_p1 = 1;
                // If the candidate is closer than the pre-computed furthest point,
                // switch them
                #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                if(candidate_dist_val[wid] < max_dist_val[p1]){
                #else
                if(candidate_dist_val < max_dist_val[p1]){
                #endif
                    if(lane == 0){
                        knn_indices[max_position[p1]] = real_p2;
                        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                        knn_sqr_dist[max_position[p1]] = candidate_dist_val[wid];
                        #else
                        knn_sqr_dist[max_position[p1]] = candidate_dist_val;
                        #endif
                    }

                    // Also update the furthest point that will be used in the next
                    // comparison
                    knn_id = real_p1*K;

                    local_max_position = -1;
                    local_max_dist = -1.0f;
                    for(j=lane; j < K; j+=32){
                        if(knn_sqr_dist[knn_id+j] > local_max_dist){
                            local_max_position = knn_id+j;
                            local_max_dist = knn_sqr_dist[knn_id+j];
                        }
                    }
                    
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  16); // assuming warpSize=32
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  16); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  8); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  8); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  4); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  4); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  2); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  2); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist, 1); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position, 1); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }

                    if(lane == 0){
                        max_dist_val[p1] = local_max_dist;
                        max_position[p1] = local_max_position;
                    }
                }
                
                if(lane == 0) atomicExch(&lock_point[p1], 0);
                // lock_point[p1] = 0;
            }

            // if(!done_p2 && !atomicOr(&lock_point[p2], 1)){
            if(!done_p2 && __any_sync(0xffffffff,!atomicCAS(&lock_point[p2], 0, 1))){
                done_p2 = 1;
                // If the candidate is closer than the pre-computed furthest point,
                // switch them
                #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                if(candidate_dist_val[wid] < max_dist_val[p2]){
                #else
                if(candidate_dist_val < max_dist_val[p2]){
                #endif
                    if(lane == 0){
                        knn_indices[max_position[p2]] = real_p1;
                        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                        knn_sqr_dist[max_position[p2]] = candidate_dist_val[wid];
                        #else
                        knn_sqr_dist[max_position[p2]] = candidate_dist_val;
                        #endif
                    }

                    // Also update the furthest point that will be used in the next
                    // comparison
                    knn_id = real_p2*K;

                    local_max_position = -1;
                    local_max_dist = -1.0f;
                    for(j=lane; j < K; j+=32){
                        if(knn_sqr_dist[knn_id+j] > local_max_dist){
                            local_max_position = knn_id+j;
                            local_max_dist = knn_sqr_dist[knn_id+j];
                        }
                    }
                    
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  16); // assuming warpSize=32
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  16); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  8); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  8); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  4); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  4); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist,  2); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position,  2); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }
                    tmp_max_dist = __shfl_down_sync( 0xffffffff, local_max_dist, 1); // assuming warpSize=32
                    tmp_max_position = __shfl_down_sync( 0xffffffff, local_max_position, 1); // assuming warpSize=32
                    if(tmp_max_dist > local_max_dist){
                        local_max_dist = tmp_max_dist;
                        local_max_position = tmp_max_position;
                    }

                    if(lane == 0){
                        max_dist_val[p2] = local_max_dist;
                        max_position[p2] = local_max_position;
                    }
                }
                if(lane == 0) atomicExch(&lock_point[p2], 0);
                // lock_point[p2] = 0;
            }
        } //end if(done p2)
        // __syncwarp();
    } // end while(not done)
}

// Assign a bucket (leaf in the tree) to each warp and a point to each thread (persistent kernel)
__global__
void compute_knn_from_buckets_pertile_coalesced_symmetric(int* points_parent,
                              int* points_depth,
                              int* accumulated_nodes_count,
                              typepoints* points,
                              int* node_idx_to_leaf_idx,
                              int* nodes_bucket,
                              int* bucket_size,
                              int* knn_indices,
                              typepoints* knn_sqr_dist,
                              int N, int D, int max_bucket_size, int K,
                              int MAX_TREE_CHILD, int total_buckets)
{
    int cbs; // cbs = current bucket size
    
    int knn_id;
    int wid = threadIdx.x / 32; // my id on warp
    int lane = threadIdx.x % 32; // my id on warp
    
    // extern __shared__ typepoints local_candidate_dist_val[];

    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
    __shared__ typepoints candidate_dist_val[32];
    #else
    typepoints candidate_dist_val;
    #endif

    int bid, p1, p2, real_p1, real_p2, _p, i, j;
    
    __shared__ int sm_leaf_bucket[300];
    __shared__ typepoints max_dist_val[300];
    __shared__ int max_position[300];

    int done_p1, done_p2;
    __shared__ int lock_point[300];
    
    bid = blockIdx.x;
    cbs = bucket_size[bid];
    for(i=threadIdx.x; i < cbs; i+=blockDim.x){
        p1 = nodes_bucket[bid*max_bucket_size + i];
        sm_leaf_bucket[i] = p1;
        lock_point[i] = 0;

        knn_id = p1*K;

        max_position[i] = knn_id;
        max_dist_val[i] = knn_sqr_dist[knn_id];
        // Finds the index of the furthest point from the current result of knn_indices
        // and the distance between them
        for(j=1; j < K; ++j){
            if(knn_sqr_dist[knn_id+j] > max_dist_val[i]){
                max_position[i] = knn_id+j; // The initial point is not necessarily in the bucket
                max_dist_val[i] = knn_sqr_dist[knn_id+j];
            }
        }

    }

    __syncthreads();
    
    for(_p = wid; _p < (cbs*cbs - cbs)/2; _p+=blockDim.x/32){
        p1 = cbs - 2 - floor(sqrt((float)((-8*_p + 4*cbs*(cbs-1)-7)))/2.0 - 0.5);
        p2 = _p + p1 + 1 - cbs*(cbs-1)/2 + (cbs-p1)*((cbs-p1)-1)/2;
        real_p1 = sm_leaf_bucket[p1];
        real_p2 = sm_leaf_bucket[p2];
        // __syncwarp();


        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        candidate_dist_val[wid] = 0.0f;
        #endif

        // __syncwarp();
        // __syncthreads();
        __syncwarp();
        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
        euclidean_distance_sqr_coalesced(real_p1,
                                         real_p2,
                                         points, D, N,
                                         lane,
                                         &candidate_dist_val[wid]);
        #else
        candidate_dist_val = euclidean_distance_sqr_coalesced(real_p1,
                                                              real_p2,
                                                              points, D, N, lane);
        #endif
        __syncwarp();
        
        if(lane == 0){
            #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
            done_p1 = candidate_dist_val[wid] >= max_dist_val[p1];
            done_p2 = candidate_dist_val[wid] >= max_dist_val[p2];
            #else
            done_p1 = candidate_dist_val >= max_dist_val[p1];
            done_p2 = candidate_dist_val >= max_dist_val[p2];
            #endif

            for(j=0; j < K && (!done_p1 || !done_p2); ++j){
                done_p1 |= real_p2 == knn_indices[real_p1*K+j];
                done_p2 |= real_p1 == knn_indices[real_p2*K+j];
            }

            while(!done_p1 || !done_p2){
                // if(!done_p1 && !atomicOr(&lock_point[p1], 1)){
                if(!done_p1 && !atomicCAS(&lock_point[p1], 0, 1)){
                    done_p1 = 1;
                    // If the candidate is closer than the pre-computed furthest point,
                    // switch them
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    if(candidate_dist_val[wid] < max_dist_val[p1]){
                    #else
                    if(candidate_dist_val < max_dist_val[p1]){
                    #endif
                        // if(real_p1 == 500) printf("%d %d %d %f %f %d\n",max_position[p1], max_position[p1]/K, max_position[p1] %K, max_dist_val[p1], candidate_dist_val[wid], real_p2);

                        knn_indices[max_position[p1]] = real_p2;
                        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                        knn_sqr_dist[max_position[p1]] = candidate_dist_val[wid];
                        #else
                        knn_sqr_dist[max_position[p1]] = candidate_dist_val;
                        #endif

                        // Also update the furthest point that will be used in the next
                        // comparison
                        knn_id = real_p1*K;
                        max_position[p1] = knn_id;
                        max_dist_val[p1] = knn_sqr_dist[knn_id];
                        for(j=1; j < K; ++j){
                            if(knn_sqr_dist[knn_id+j] > max_dist_val[p1]){
                                max_position[p1] = knn_id+j;
                                max_dist_val[p1] = knn_sqr_dist[knn_id+j];
                            }
                        }
                    }
                    atomicExch(&lock_point[p1], 0);
                    // lock_point[p1] = 0;
                }

                // if(!done_p2 && !atomicOr(&lock_point[p2], 1)){
                if(!done_p2 && !atomicCAS(&lock_point[p2], 0, 1)){
                    done_p2 = 1;
                    // If the candidate is closer than the pre-computed furthest point,
                    // switch them
                    #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                    if(candidate_dist_val[wid] < max_dist_val[p2]){
                    #else
                    if(candidate_dist_val < max_dist_val[p2]){
                    #endif
                        knn_indices[max_position[p2]] = real_p1;
                        #if EUCLIDEAN_DISTANCE_VERSION!=EDV_NOATOMIC_NOSHM && EUCLIDEAN_DISTANCE_VERSION!=EDV_WARP_REDUCE_XOR_NOSHM
                        knn_sqr_dist[max_position[p2]] = candidate_dist_val[wid];
                        #else
                        knn_sqr_dist[max_position[p2]] = candidate_dist_val;
                        #endif
        
                        // Also update the furthest point that will be used in the next
                        // comparison
                        knn_id = real_p2*K;
                        max_position[p2] = knn_id;
                        max_dist_val[p2] = knn_sqr_dist[knn_id];
                        for(j=1; j < K; ++j){
                            if(knn_sqr_dist[knn_id+j] > max_dist_val[p2]){
                                max_position[p2] = knn_id+j;
                                max_dist_val[p2] = knn_sqr_dist[knn_id+j];
                            }
                        }
                    }
                    atomicExch(&lock_point[p2], 0);
                    // lock_point[p2] = 0;
                }
            }
        }
        __syncwarp();
    }
    
}


#endif