#include "hip/hip_runtime.h"
/*
This file is part of the GPU-RSFK Project (https://github.com/BrunoMeyer/gpu-rsfk).

BSD 3-Clause License

Copyright (c) 2021, Bruno Henrique Meyer, Wagner M. Nunan Zola
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef __BUILD_TREE_UPDATE_PARENTS_ANN__CU
#define __BUILD_TREE_UPDATE_PARENTS_ANN__CU

#include "../include/common.h"

__global__
void
build_tree_utils_ann(int* actual_depth,
                 int* depth_level_count,
                 int* count_new_nodes,
                 int* tree_count,
                 int* accumulated_nodes_count,
                 int* device_active_points_count)
// Executed at end of each iteration of tree building
{
    *actual_depth = *actual_depth+1;
    *device_active_points_count = 0;
}

__global__
void build_tree_update_parents_ann(
    RSFK_typepoints* tree,
    int* tree_parents,
    int* tree_children,
    int* points_parent,
    int* points_depth,
    int* is_right_child,
    bool* is_leaf,
    bool* is_leaf_new_depth,
    int* child_count,
    int* child_count_new_depth,
    RSFK_typepoints* points,
    int* actual_depth,
    int* tree_count,
    int* depth_level_count,
    int* count_new_nodes,
    int N, int D,
    int MIN_TREE_CHILD, int MAX_TREE_CHILD)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;

    int right_child, p;
    int updated_count;
    // Set nodes parent in the new depth
    for(p = tid; p < N; p+=blockDim.x*gridDim.x){
        if(points_depth[p] == *actual_depth-1 && !is_leaf[points_parent[p]]){
            right_child = is_right_child[p];
            points_parent[p] = tree_children[2*points_parent[p]+right_child];
            points_depth[p] = *actual_depth;
            updated_count = atomicAdd(&child_count_new_depth[points_parent[p]],1)+1;
        }
    }
}

#endif