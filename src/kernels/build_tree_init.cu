#include "hip/hip_runtime.h"
/*
This file is part of the GPU-RSFK Project (https://github.com/BrunoMeyer/gpu-rsfk).

BSD 3-Clause License

Copyright (c) 2021, Bruno Henrique Meyer, Wagner M. Nunan Zola
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef __BUILD_TREE_INIT__CU
#define __BUILD_TREE_INIT__CU

#include "../include/common.h"

__global__
void build_tree_init(RSFK_typepoints* tree,
                     int* tree_parents,
                     int* tree_children,
                     int* points_parent,
                     int* child_count,
                     bool* is_leaf,
                     RSFK_typepoints* points,
                     int* actual_depth,
                     int* tree_count,
                     int* depth_level_count,
                     int* accumulated_nodes_count,
                     int* accumulated_child_count,
                     int* device_count_points_on_leafs,
                     int* count_new_nodes,
                     int N, int D, int RANDOM_SEED)
{
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t r; 
    hiprand_init(RANDOM_SEED+tid, // the seed controls the sequence of random values that are produced
            blockIdx.x,  // the sequence number is only important with multiple cores 
            tid,  // the offset is how much extra we advance in the sequence for each call, can be 0 
            &r);

    int p1, p2;
    // Sample two random points
    if(tid == 0){
        p1 = hiprand(&r) % N;
        p2 = p1;
        // Ensure that two different points was sampled
        while(p1 == p2 && N > 1){
            p2 = hiprand(&r) % N;
        }
        *count_new_nodes = 1;
        
        create_root(tree, tree_parents, tree_children, tree_count, p1, p2,
                    count_new_nodes, points, D, N);
        depth_level_count[0] = 1;
        accumulated_nodes_count[0] = 0;
        is_leaf[0] = false;
        child_count[0] = N;
        accumulated_child_count[0] = 0;
        *actual_depth = 1;
    }
}

#endif