#include "hip/hip_runtime.h"
/*
This file is part of the GPU-RSFK Project (https://github.com/BrunoMeyer/gpu-rsfk).

BSD 3-Clause License

Copyright (c) 2021, Bruno Henrique Meyer, Wagner M. Nunan Zola
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef __CREATE_NODE__CU
#define __CREATE_NODE__CU

#include "../include/common.h"

__device__
inline
void create_root(RSFK_typepoints* tree,
                 int* tree_parents,
                 int* tree_children,
                 int* tree_count,
                 int p1,
                 int p2,
                 int* count_new_nodes,
                 RSFK_typepoints* points,
                 int D, int N)
{
    // Average point
    // node_path*D*2 : D*2 = size of centroid point and normal vector

    int node_idx = 0;
    
    tree_parents[node_idx] = -1;
    *tree_count = 0;

    int i;
    // tree[node_idx*(D+1) + D] = 0.0f;
    tree[get_tree_idx(node_idx,D,*count_new_nodes,D)] = 0.0f;

    for(i=0;i < D; ++i){
        tree[get_tree_idx(node_idx,i,*count_new_nodes,D)] = points[get_point_idx(p1,i,N,D)]-points[get_point_idx(p2,i,N,D)];
        tree[get_tree_idx(node_idx,D,*count_new_nodes,D)]+= tree[get_tree_idx(node_idx,i,*count_new_nodes,D)]*(points[get_point_idx(p1,i,N,D)]+points[get_point_idx(p2,i,N,D)])/2; // multiply the point of plane and the normal vector 
    }
}

__device__
inline
void create_node(int parent,
                 int is_right_child,
                 RSFK_typepoints* tree,
                 int* tree_parents,
                 int* tree_children,
                 int* tree_count,
                 int* count_new_nodes,
                 int p1,
                 int p2,
                 RSFK_typepoints* points,
                 int D, int N)
{
    // Average point
    // node_path*D*2 : D*2 = size of centroid point and normal vector

    int node_idx = atomicAdd(tree_count, 1);
    tree_parents[node_idx] = parent;
    
    tree_children[2*parent+is_right_child] = node_idx;
    int i;

    RSFK_typepoints s = 0.0f;
    for(i=0; i < D; ++i){
        tree[get_tree_idx(node_idx,i,*count_new_nodes,D)] = points[get_point_idx(p1,i,N,D)]-points[get_point_idx(p2,i,N,D)];
        s+= tree[get_tree_idx(node_idx,i,*count_new_nodes,D)]*(points[get_point_idx(p1,i,N,D)]+points[get_point_idx(p2,i,N,D)])/2; // multiply the point of plane and the normal vector 
    }
    tree[get_tree_idx(node_idx,D,*count_new_nodes,D)] = s;
}

#endif